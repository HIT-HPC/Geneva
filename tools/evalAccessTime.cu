#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <ctime>
#include <map>
#include <vector>
#include <queue>
#include <algorithm>

using namespace std;

#define INVALID 0xffffffff
#define HASHSEED 17
typedef int LABEL;
typedef int VID;
typedef int EID;
typedef int GID;
typedef long PID;
typedef long LENGTH;
class Neighbor
{
public:
	VID vid;
	LABEL elb;
	Neighbor()
	{
		vid = -1;
		elb = -1;
	}
	Neighbor(int _vid, int _elb)
	{
		vid = _vid;
		elb = _elb;
	}
	bool operator<(const Neighbor& _nb) const
	{
		if(this->elb == _nb.elb)
		{
			return this->vid < _nb.vid;
		}
		else
		{
			return this->elb < _nb.elb;
		}
	}
};

class Element
{
public:
	int label;
	int id;
	bool operator<(const Element& _ele) const
	{
		if(this->label == _ele.label)
		{
			return this->id <_ele.id;
		}
		else
		{
			return this->label < _ele.label;
		}
	}
};

class Vertex
{
public:
	//VID id;
	LABEL label;
	//NOTICE:VID and EID is just used in this single graph
	vector<Neighbor> in;
	vector<Neighbor> out;
	Vertex()
	{
		label = -1;
	}
	Vertex(LABEL lb):label(lb)
	{
	}
};

class PCSR
{
public:
    unsigned* row_offset;  //the size is 32*key_num
    unsigned* column_index;
    unsigned key_num;  //also the group number
    unsigned edge_num;
    PCSR()
    {
        row_offset = NULL;
        column_index = NULL;
        key_num = 0;
        edge_num = 0;
    }
    ~PCSR()
    {
        delete[] row_offset;
        delete[] column_index;
    }
    inline unsigned getEdgeNum() const
    {
        return this->edge_num;
    }
};
uint32_t MurmurHash2CPU(const void * key, int len, uint32_t seed) 
{
    const uint32_t m = 0x5bd1e995;
    const int r = 24;
    // Initialize the hash to a 'random' value
    uint32_t h = seed ^ len;
    // Mix 4 bytes at a time into the hash
    const unsigned char * data = (const unsigned char *) key;
    while (len >= 4) 
    {
        uint32_t k = *(uint32_t*) data;
        k *= m;
        k ^= k >> r;
        k *= m;
        h *= m;
        h ^= k;
        data += 4;
        len -= 4;
    }
    // Handle the last few bytes of the input array
    switch (len) 
    {
        case 3:
            h ^= data[2] << 16;
        case 2:
            h ^= data[1] << 8;
        case 1:
          h ^= data[0];
          h *= m;
    };
    // Do a few final mixes of the hash to ensure the last few
    // bytes are well-incorporated.
    h ^= h >> 13;
    h *= m;
    h ^= h >> 15;
    return h;
}
class Graph
{
public:
	std::vector<Vertex> vertices;
	void addVertex(LABEL _vlb){
		this->vertices.push_back(Vertex(_vlb));
	}
	void addEdge(VID _from, VID _to, LABEL _elb){
		this->vertices[_from].out.push_back(Neighbor(_to, _elb));
		this->vertices[_to].in.push_back(Neighbor(_from, _elb));
	}

    	unsigned vertexLabelNum, edgeLabelNum;
	//CSR format: 4 pointers
	unsigned vertex_num;
	unsigned* vertex_value;

    	PCSR* csrs_in;
    	PCSR* csrs_out;

	Graph() 
	{
		vertex_num = 0;
        	csrs_in = csrs_out = NULL;
	}
	~Graph() 
	{ 
		delete[] vertex_value;
        	delete[] csrs_in;
        	delete[] csrs_out;
	}

    	void buildPCSR(PCSR* pcsr, std::vector<unsigned>& keys, int label, bool incoming){
		unsigned key_num = keys.size();
    		unsigned* row_offset = new unsigned[key_num * 32];
    		unsigned edge_num = pcsr->edge_num;
    		unsigned* column_index = new unsigned[edge_num];
    		pcsr->key_num = key_num;
    		pcsr->row_offset = row_offset;
    		pcsr->column_index = column_index;
    		for(int i = 0; i < key_num*16; ++i)
    		{
    		    row_offset[2*i] = INVALID;
    		    row_offset[2*i+1] = 0;
    		}
    		for(int i = 0; i < edge_num; ++i)
    		{
    		    column_index[i] = INVALID;
    		}
    		vector<unsigned>* buckets = new vector<unsigned>[key_num];
    		for(int i = 0; i < key_num; ++i)
    		{
    		    unsigned id = keys[i];
    		    unsigned pos = MurmurHash2CPU(&id, 4, HASHSEED) % key_num;
    		    buckets[pos].push_back(id);
    		}
    		queue<unsigned> empty_buckets;
    		for(int i = 0; i < key_num; ++i)
    		{
    		    if(buckets[i].empty())
    		    {
    		        empty_buckets.push(i);
    		    }
    		}
    		for(int i = 0; i < key_num; ++i)
    		{
    		    if(buckets[i].empty())
    		    {
    		        continue;
    		    }
    		    int tsize = buckets[i].size(), j;
    		    if(tsize > 15)
    		    {
    		        cout<<"DETECTED: more than 1 buckets are needed!"<<endl;
    		        exit(1);
    		    }
    		    else if(tsize > 30)
    		    {
    		        cout<<"DETECTED: more than 2 buckets are needed!"<<endl;
    		        exit(1);
    		    }
    		    for(j = 0; j < 15 && j < tsize; ++j)
    		    {
    		        row_offset[32*i+2*j] = buckets[i][j];
    		    }
    		    if(j < tsize)
    		    {
    		        int another_bucket = empty_buckets.front(), k = 0;
    		        empty_buckets.pop();
    		        row_offset[32*i+30] = another_bucket;
    		        while(j < tsize)
    		        {
    		            row_offset[32*another_bucket+2*k] = buckets[i][j];
    		            ++j;
    		            ++k;
    		        }
    		    }
    		}
    		delete[] buckets;

    		unsigned pos = 0;
    		for(int i = 0; i < key_num; ++i)
    		{
    		    int j;
    		    for(j = 0; j < 15; ++j)
    		    {
    		        unsigned id = row_offset[32*i+2*j];
    		        if(id == INVALID)
    		        {
    		            break;
    		        }
    		        vector<Neighbor>* adjs = &this->vertices[id].out;
    		        if(incoming)
    		        {
    		            adjs = &this->vertices[id].in;
    		        }
    		        row_offset[32*i+2*j+1] = pos;
    		        for(int k = 0; k < adjs->size(); ++k)
    		        {
    		            if((*adjs)[k].elb == label)
    		            {
    		                column_index[pos++] = (*adjs)[k].vid;
    		            }
    		        }
    		    }
    		    //set final next offset in this group, also the start offset of next valid ID
    		    row_offset[32*i+2*j+1] = pos;
    		    //row_offset[32*i+31] = pos;
    		}
	}
	void transformToCSR(){
		this->vertex_num = this->vertices.size();
		this->vertex_value = new unsigned[this->vertex_num];
		for(int i = 0; i < this->vertex_num; ++i)
		{
			this->vertex_value[i] = this->vertices[i].label;
        		sort(this->vertices[i].in.begin(), this->vertices[i].in.end());
        		sort(this->vertices[i].out.begin(), this->vertices[i].out.end());
    		}

    		//NOTICE: the edge label begins from 1
    		this->csrs_in = new PCSR[this->edgeLabelNum+1];
    		this->csrs_out = new PCSR[this->edgeLabelNum+1];
    		vector<unsigned>* keys_in = new vector<unsigned>[this->edgeLabelNum+1];
    		vector<unsigned>* keys_out = new vector<unsigned>[this->edgeLabelNum+1];
		for(int i = 0; i < this->vertex_num; ++i)
    		{
        		int insize = this->vertices[i].in.size(), outsize = this->vertices[i].out.size();
        		for(int j = 0; j < insize; ++j)
        		{
        		    	int vid = this->vertices[i].in[j].vid;
        		    	int elb = this->vertices[i].in[j].elb;
        		    	int tsize = keys_in[elb].size();
        		    	if(tsize == 0 || keys_in[elb][tsize-1] != i)
        		    	{
        		    	    keys_in[elb].push_back(i);
        		    	}
        		    	//NOTICE: we do not use C++ reference PCSR& here because it can not change(frpm p-->A to p-->B)
        		    	PCSR* tcsr = &this->csrs_in[elb];
        		    	tcsr->edge_num++;
        		}
        		for(int j = 0; j < outsize; ++j)
        		{
        		    int vid = this->vertices[i].out[j].vid;
        		    int elb = this->vertices[i].out[j].elb;
        		    int tsize = keys_out[elb].size();
			    // cout<<tsize<<endl;
        		    // for (const auto &c : keys_out[elb]) cout << c << " ";
        		    if(tsize == 0 || keys_out[elb][tsize-1] != i)
        		    {
        		        keys_out[elb].push_back(i);
        		    }
        		    PCSR* tcsr = &this->csrs_out[elb];
        		    tcsr->edge_num++;
        		}
    		}

    		for(int i = 1; i <= this->edgeLabelNum; ++i)
    		{
    		    	PCSR* tcsr = &this->csrs_in[i];
    		    	this->buildPCSR(tcsr, keys_in[i], i, true);
    		    	tcsr = &this->csrs_out[i];
    		    	this->buildPCSR(tcsr, keys_out[i], i, false);
    		}
    		delete[] keys_in;
    		delete[] keys_out;
	}

	inline unsigned vSize() const
	{
		return vertex_num;
	}
};



Graph* input(FILE* fp)
{
	char c1, c2;
	int id0, id1, id2, lb;
	bool flag = false;
	Graph* ng = NULL;

	while(true)
	{
		fscanf(fp, "%c", &c1);
		if(c1 == 't')
		{
			if(flag)
			{
				fseek(fp, -1, SEEK_CUR);
				return ng;
			}
			flag = true;
			fscanf(fp, " %c %d\n", &c2, &id0);
			if(id0 == -1)
			{
				return NULL;
			}
			else
			{
				ng = new Graph;
			}
			//read vertex num, edge num, vertex label num, edge label num
			int numVertex, numEdge;
			fscanf(fp, " %d %d %d %d\n", &numVertex, &numEdge, &(ng->vertexLabelNum), &(ng->edgeLabelNum));
		}
		else if(c1 == 'v')
		{
			fscanf(fp, " %d %d\n", &id1, &lb);
			ng->addVertex(lb); 
		}
		else if(c1 == 'e')
		{
			fscanf(fp, " %d %d %d\n", &id1, &id2, &lb);
			//NOTICE:we treat this graph as directed, each edge represents two
			//This may cause too many matchings, if to reduce, only add the first one
			//ng->addEdge(id1, id2, lb+1);
			ng->addEdge(id1, id2, lb);
			//ng->addEdge(id2, id1, lb);
		}
		else if (c1 == '#')
		{
			char tmpc[100];
           		fscanf(fp, "%[^\n]%*c", tmpc);
			continue;
		}
		else 
		{
			cerr<<"ERROR in input() -- invalid char"<<endl;
			return NULL;
		}
	}
	return NULL;
}

__device__ uint32_t 
MurmurHash2GPU(const void * key, int len, uint32_t seed) 
{
    const uint32_t m = 0x5bd1e995;
    const int r = 24;
    // Initialize the hash to a 'random' value
    uint32_t h = seed ^ len;
    // Mix 4 bytes at a time into the hash
    const unsigned char * data = (const unsigned char *) key;
    while (len >= 4) 
    {
        uint32_t k = *(uint32_t*) data;
        k *= m;
        k ^= k >> r;
        k *= m;
        h *= m;
        h ^= k;
        data += 4;
        len -= 4;
    }
    // Handle the last few bytes of the input array
    switch (len) 
    {
        case 3:
            h ^= data[2] << 16;
        case 2:
            h ^= data[1] << 8;
        case 1:
          h ^= data[0];
          h *= m;
    };
    // Do a few final mixes of the hash to ensure the last few
    // bytes are well-incorporated.
    h ^= h >> 13;
    h *= m;
    h ^= h >> 15;
    return h;
}

//searchnum should be a multiple of 32
__global__ void
join_kernel(unsigned* row_offset, unsigned *result, unsigned long keynum, unsigned long searchnum)
{
    	__shared__ unsigned s_pool1[256];
    	__shared__ unsigned s_pool3[256];
	unsigned i,laneId=threadIdx.x;
	unsigned valid=0,invalid=0;
	unsigned tmp1 = result[laneId];
	unsigned tmp2 = result[laneId+32];
    	unsigned idx = threadIdx.x & 0x1f;
    	const uint32_t m = 0x5bd1e995;
    	const int r = 24;
	for(i=1;i<=searchnum;i++){
    		unsigned bgroup = threadIdx.x & 0xffffffe0;  //equal to (x/32)*32
		unsigned vid = i;
    		unsigned bucket;// = MurmurHash2GPU(&vid, 4, HASHSEED) % keynum;

    		uint32_t h = HASHSEED ^ 4;
    		const unsigned char * data = (const unsigned char *) (&vid);
        	uint32_t k = *(uint32_t*) data;
        	k *= m;
        	k ^= k >> r;
        	k *= m;
        	h *= m;
        	h ^= k;
        	data += 4;
    		h ^= h >> 13;
    		h *= m;
    		h ^= h >> 15;
    		bucket= (h&4095);
    		s_pool1[bgroup+idx] = row_offset[32*bucket+idx];
    		if(idx == 0)
    		{
    		    s_pool3[bgroup] = INVALID;
    		}
    		if(idx < 30 && (idx&1)==0)
    		{
    		    if(s_pool1[bgroup+idx] == i)
    		    {
    		        s_pool3[bgroup] = s_pool1[bgroup+idx+1];
    		        s_pool3[bgroup+1] = s_pool1[bgroup+idx+3];
    		    }
    		}
    		if(s_pool3[bgroup] == INVALID)  // not found
    		{
    		    invalid++;
    		}
		else{
			valid++;
		}
	}
	result[laneId] = tmp1 + valid;
	result[laneId+32] = tmp2 + invalid;
}

__global__ void init(unsigned *result){
	unsigned i=blockIdx.x*blockDim.x+threadIdx.x;
	result[i] = 0;
}

void loadDataBase(string filename, string metafilename, map<uint,uint*> &labelPartitions){
    ifstream graph(filename,ios::in|ios::binary);
    if(!graph.is_open()){
        cout<<"open "<<filename <<" wrong"<<endl;
        exit(0);
    }
    ifstream metafile(metafilename,ios::in);
    if(!metafile.is_open()){
        cout<<"open metafile wrong"<<endl;
        exit(0);
    }
    map<uint,uint> v_label;
    uint oldvid,label,newvid,maxid=0;
    while(metafile>>newvid>>label>>oldvid){
        v_label.insert(pair<uint,uint>(newvid,label));
        if(newvid>maxid){
            maxid = newvid;
        }
    }
    metafile.close();
    //cout<<"maxid="<<maxid<<endl;
    //i don't know how many bytes are aligned, but 16 uints should be fine
    uint allocsizev = (((maxid+1+15)>>4)<<4);
    uint *vertex_label = (uint*)malloc(sizeof(uint)*allocsizev);
    vertex_label[0] = maxid;
	//cout<<"in loaddatabase"<<endl;
    map<uint,uint>::iterator ite;
    for(ite=v_label.begin();ite!=v_label.end();++ite){
        uint id = ite->first;
        uint lid = ite->second;
        vertex_label[id] = lid;
    }
	//cout<<"after reading v label"<<endl;
    uint tmp;
    uint j=1,index=0;
    graph.read((char*)&tmp,sizeof(uint));
    uint totpart = tmp;
	//cout<<"in loadDatabase, totpart="<<totpart<<endl;
    while(index<totpart){
        graph.read((char*)&tmp,sizeof(uint));
        uint totLabelPartEleNum = tmp;
	//cout<<"part "<<index+1<<" num="<<totLabelPartEleNum<<endl;
        uint allocsize = (((totLabelPartEleNum+15)>>4)<<4);
        uint *tmpuint = new uint[allocsize];
        graph.read((char*)&tmpuint[1],sizeof(uint)*(totLabelPartEleNum-1));
        tmpuint[0] = totLabelPartEleNum;
        labelPartitions.insert(pair<uint,uint*>(j,tmpuint));
        ++j;
        ++index;
    }
    labelPartitions.insert(pair<uint,uint*>(0xffffffff,vertex_label));
    graph.close();
}


__global__ void accessTimeMy(unsigned* row_offset, unsigned *result, unsigned intervalNum, unsigned long searchnum) {

	uint laneId, i, j;
    	laneId = threadIdx.x & 31;
	unsigned tmp1 = result[laneId];
	unsigned tmp2 = result[laneId+32];
    	__shared__ uint indexForIndex[256*2+1+1+64+1];
    	for(i=threadIdx.x;i<intervalNum*2+1+1;i=i+32){
        	indexForIndex[i] = row_offset[i];
    	}
	uint valid=0, invalid=0;
    	if(laneId==0){ indexForIndex[256*2+66] = 0; }
	row_offset = row_offset+intervalNum*2+1+1;
    	for(i=1;i<searchnum;++i) {
        	uint svid = i;
        	uint lowerLimit = 0, upperLimit, predicate = 0, index;
        	uint len = (intervalNum+31)&0xffffffe0;
        	for(j=laneId;j<len;j=j+32){
            		lowerLimit = indexForIndex[j];
            		upperLimit = indexForIndex[j+intervalNum+1];
            		upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            		predicate = j<intervalNum?(svid>=lowerLimit && svid<upperLimit):0;
            		predicate = __ballot_sync(0xffffffff,predicate);
            		if(predicate>0){
                		index = svid-lowerLimit+indexForIndex[j+intervalNum];
                		uint tmpIndex = __ffs(predicate)-1;
                		index = __shfl_sync(0xffffffff,index,tmpIndex);
                		break;
            		}
        	}
		uint pos = row_offset[index];
        	if(predicate==0){ 
        	  	invalid = pos+invalid;
        	}else{
			valid = pos+valid;
		}
    	}
	result[laneId] = tmp1+valid;
	result[laneId+32] = tmp2+invalid;
}

int main(int argc, const char *argv[])
{
    	hipError_t err;
	int labelnum;
	unsigned long totnum = 0, totkeynum=0;
	int *pos;
	unsigned *keynum;
	string data = argv[1];
	string inputFileName = "/WORKSPACE/lgz/subgraph_matching/subgraph_matching/datasets/"+data+".mygraph";
	string metaFileName = "/WORKSPACE/lgz/subgraph_matching/subgraph_matching/datasets/"+data+".metadata";
	data = "/WORKSPACE/lgz/subgraph_matching/GSI/data/"+data+"/"+data+".g";
	FILE *fp = fopen(data.c_str(),"r");
	if(fp==NULL){
		cout<<"can not open file "<<data<<endl;
		return 0;
	}
	Graph *datagraph = NULL;
	datagraph = input(fp);
	if(datagraph == NULL){
		cout<<"wrong data graph"<<endl;
		return 0;
	}
	datagraph->transformToCSR();
	labelnum = datagraph->edgeLabelNum;
	cout<<"construct pcsr done, GSI has "<<labelnum<<" edge labels"<<endl;
	pos = new int[labelnum+1];
	keynum = new unsigned[labelnum+1];
	for(int label=1;label<=labelnum;++label){
		PCSR* tcsr;
        	tcsr = &(datagraph->csrs_out[label]);
		pos[label] = totnum;
		totnum = totnum+tcsr->key_num*32;
		keynum[label] = tcsr->key_num;
		totkeynum = totkeynum + tcsr->key_num;
	}

	unsigned *alldata;
    	hipMalloc(&alldata, sizeof(unsigned)*totnum);
	for(int label=1;label<=labelnum;++label){
		PCSR* tcsr;
        	tcsr = &(datagraph->csrs_out[label]);
		totnum = tcsr->key_num*32;
		unsigned *dev = alldata+pos[label];
    		hipMemcpy(dev, tcsr->row_offset, totnum*sizeof(unsigned), hipMemcpyHostToDevice);
	}
	unsigned searchnum = ((datagraph->vertex_num)/64)*64;
	cout<<"searchnum="<<searchnum<<endl;
	unsigned numBlocks=1;
	unsigned *result;
	hipMalloc(&result,sizeof(unsigned)*numBlocks*64);
	init<<<numBlocks,64>>>(result);
    	hipDeviceSynchronize();
    
	double start = clock();
	for(int label=1;label<=labelnum;++label){
		unsigned *dev = alldata+pos[label];
		join_kernel<<<numBlocks, 32>>>(dev, result, keynum[label], searchnum);
	}
    	hipDeviceSynchronize();
	double end = clock();
    	double endtime=(double)(end-start)/CLOCKS_PER_SEC;
    	std::cout<<"GSI time: "<<endtime*1000<<std::endl;
    	err = hipGetLastError();
    	std::cout<<hipGetErrorString(err)<<std::endl;
	unsigned *resulthost = (unsigned *)malloc(sizeof(unsigned)*numBlocks*64);
    	hipMemcpy(resulthost, result, numBlocks*64*sizeof(unsigned), hipMemcpyDeviceToHost);
	cout<<"valid = "<<resulthost[0]<< " invalid = "<< resulthost[32]<<endl;
	hipFree(alldata);



    	map<unsigned,unsigned *> allEdgeLabelPartitions;
	loadDataBase(inputFileName, metaFileName,allEdgeLabelPartitions);
	cout<<"my has "<<allEdgeLabelPartitions.size()-1<<" edge labels"<<endl;
	unsigned *intervalNum = new unsigned[labelnum+1];
	unsigned *posmy = new unsigned[labelnum+1];
	unsigned totlen=0, len;
        for(int label=1;label<=labelnum;++label){
		posmy[label] = totlen;
		unsigned *edgeLabelPartition = allEdgeLabelPartitions[label];
        	len = edgeLabelPartition[0]-5;
		totlen = len+totlen;
        	intervalNum[label] = edgeLabelPartition[2];
	}
	unsigned *datadev;
	hipMalloc(&datadev,sizeof(unsigned)*totlen);
	for(int label=1;label<=labelnum;++label){
		unsigned *edgeLabelPartition = allEdgeLabelPartitions[label];
        	len = edgeLabelPartition[0]-5;
        	hipMemcpy(datadev+posmy[label],edgeLabelPartition+5,sizeof(unsigned)*len,hipMemcpyHostToDevice);
	}
	init<<<numBlocks,64>>>(result);
    	hipDeviceSynchronize();
	start = clock();
	for(int label=1;label<=labelnum;++label){
		unsigned *edgeLabelPartition = allEdgeLabelPartitions[label];
        	len = edgeLabelPartition[0]-5;
		accessTimeMy<<<1,32>>>(datadev+posmy[label],result, intervalNum[label],searchnum);
	}
    	hipDeviceSynchronize();
	end = clock();
    	endtime=(double)(end-start)/CLOCKS_PER_SEC;
    	std::cout<<"my time: "<<endtime*1000<<std::endl;
    	err = hipGetLastError();
    	std::cout<<hipGetErrorString(err)<<std::endl;
    	hipMemcpy(resulthost, result, numBlocks*64*sizeof(unsigned), hipMemcpyDeviceToHost);
	cout<<"valid = "<<resulthost[0]<<" invalid="<<resulthost[32]<<endl;
	//cudaFree(datadev);
	hipFree(result);
	free(resulthost);
}
