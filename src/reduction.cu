#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
extern int GPU_SM_NUM;
#define GETBASEADDR                                                                         \
    lowerLimit=0;                                                                           \
    predicate=0;                                                                            \
    len = (intervalNum+31)&0xffffffe0;                                                      \
    for(uint l=laneId;l<len;l=l+32){                                                             \
        lowerLimit = indexForIndex[l];                                                      \
        upperLimit = indexForIndex[l+intervalNum+1];                                        \
        upperLimit = lowerLimit+upperLimit-indexForIndex[l+intervalNum];                    \
        predicate = l<intervalNum?(svid>=lowerLimit && svid<upperLimit):0;                  \
        predicate = __ballot_sync(0xffffffff,predicate);                                    \
        if(predicate>0){                                                                    \
            index = svid-lowerLimit+indexForIndex[l+intervalNum];                           \
            uint tmpIndex = __ffs(predicate)-1;                                               \
            index = __shfl_sync(0xffffffff,index,tmpIndex);                                 \
            break;                                                                          \
        }                                                                                   \
    }                                                                                       \
    if(predicate==0){                                                                       \
        goto not_valid_nextEmb;                                                                             \
    }                                                                                       \
    baseNum = edgeLabelPartition[index];                                                    \
    baseAddr = neighborsData+baseNum;                                                       \
    baseLen = edgeLabelPartition[index+1]-baseNum;


#define GETLABEL_ADDR_LEN(pairIndex)                                                        \
    useRecord = (recordFlag>>pairIndex)&0x00000001;                                         \
    isRecord = (recordFlag>>pairIndex)&00010000;                                            \
    indexNum = 0;                                                                           \
    if(useRecord==1){                                                                       \
        uint distance = recordPos[index*2];                                                 \
        tmpNeigLen = recordPos[index*2+1];                                                  \
        if(distance==0 && tmpNeigLen==0){                                                   \
            lowerLimit = 0;                                                                 \
            upperLimit = baseLen;                                                       \
            FIND_LABEL_LIMIT(baseAddr, lowerLimit, upperLimit, evidlabel)                   \
            tmpNeigLen = upperLimit - lowerLimit;                                       \
            if (tmpNeigLen == 0) {                                                          \
                if(isRecord){                                                               \
                    if(laneId==0){ recordPos[index*2]=1; recordPos[index*2+1]=0; }          \
                }                                                                           \
                goto not_valid_nextEmb;                                                     \
            }                                                                               \
            if (tmpNeigLen > VBLOCKSIZE+2 && tmpNeigLen <= 32 * VBLOCKSIZE + 32){             \
                indexNum = (tmpNeigLen+VBLOCKSIZE) / (VBLOCKSIZE + 1);                            \
            }                                                                               \
            else if (tmpNeigLen > 32 * VBLOCKSIZE + 32) { indexNum= 32; }                   \
            tmpNeigAddr=baseAddr+lowerLimit;                                                \
            tmpNeigLen=tmpNeigLen-indexNum;                                                 \
            if (isRecord) {                                                                 \
                if (laneId == 0) {                                                          \
                    recordPos[index * 2] = lowerLimit;                                      \
                    recordPos[index * 2 + 1] = tmpNeigLen;                                  \
                }                                                                           \
            }                                                                               \
        }else if(distance==1&&tmpNeigLen==0){ goto not_valid_nextEmb; }                     \
    }else {                                                                                 \
        lowerLimit = 0; upperLimit = baseLen;                                           \
        FIND_LABEL_LIMIT(baseAddr, lowerLimit, upperLimit, evidlabel)                       \
        tmpNeigLen = upperLimit - lowerLimit;                                           \
        if(tmpNeigLen==0){                                                                  \
            if(isRecord){                                                                   \
                if(laneId==0){                                                              \
                    recordPos[index*2]=1;                                                   \
                    recordPos[index*2+1]=0;                                                 \
                }                                                                           \
            }                                                                               \
            goto not_valid_nextEmb;                                                         \
        }                                                                                   \
        if (tmpNeigLen > VBLOCKSIZE+2 && tmpNeigLen <= 32 * VBLOCKSIZE + 32) {                \
            indexNum = (tmpNeigLen+VBLOCKSIZE) / (VBLOCKSIZE + 1);                                \
        }                                                                                   \
        else if (tmpNeigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }                      \
        tmpNeigAddr = baseAddr + lowerLimit;                                                \
        tmpNeigLen = tmpNeigLen - indexNum;                                                 \
        if (isRecord) {                                                                     \
            if (laneId == 0) {                                                              \
                recordPos[index * 2] = lowerLimit;                                          \
                recordPos[index * 2 + 1] = tmpNeigLen;                                      \
            }                                                                               \
        }                                                                                   \
    }


#define FINDEVID                                                                            \
    if(indexNum>0){                                                                         \
        upperLimit = laneId<indexNum?tmpNeigAddr[laneId]:0;                                 \
        lowerLimit = __shfl_up_sync(0xffffffff,upperLimit,1);                               \
        if(laneId==0) { lowerLimit = tmpNeigAddr[indexNum]-1; }                             \
        predicate = (evid>lowerLimit && evid<=upperLimit)?1:0;                              \
        predicate = __ballot_sync(0xffffffff,predicate);                                    \
        if(predicate==0){ goto not_valid_nextEmb; }                                         \
        predicate = __ffs(predicate)-1;                                                       \
        lowerLimit = predicate*blockSize;                                                   \
        upperLimit = predicate==indexNum-1?tmpNeigLen:lowerLimit+blockSize;                 \
    }                                                                                       \
    for(uint l=lowerLimit+laneId;l<((upperLimit+31)&0xffffffe0);l=l+32){                         \
        uint tmpV = l<upperLimit?tmpNeigAddr[l]:0;                                          \
        predicate = tmpV==evid;                                                             \
        predicate = __ballot_sync(0xffffffff,predicate);                                    \
        if(predicate>0){ break; }                                                           \
    }                                                                                       \
    if(predicate==0) { goto not_valid_nextEmb; }


//the format for indexForIndex[0:2*256] is vs1,vs2,vs3,len1,len2,len3. len1 is the numberr of vertices in interval 1, len2
//is the number of vertices in all previsous intervals (including interval 2).
//the format for indexForIndex[2*256:] is totUintNum(uint),(start1,end1),(start2,end2),labels(16,we do not need svid label),recordPos(16).
// first, all same start are grouped together, in each group
//all end with same label are grouped. each number uses 5 bits, each time we at most process 16 pairs. we use one uint to represent use
//use record and isrecord.
template<bool isLastPhase>
__global__ void reductionPhase_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, uint *neighborsData,
    uint *baseRecordPos, uint *auxArray, uint *partialEmb, uint intervalNum, uint partialRowNum, uint embLen){

    uint laneId, gridWarpNum, warpIdInBlock, totUintNum, i, j;
    laneId = threadIdx.x & 31;
    gridWarpNum = (gridDim.x * blockDim.x)>>5;
    warpIdInBlock = threadIdx.x >> 5;
    __shared__ uint neigV[WARPPERBLOCK][33];
    __shared__ uint indexForIndex[256*2+1+1+64];
    for(i=threadIdx.x;i<intervalNum*2+1+1;i=i+blockDim.x){
        indexForIndex[i] = edgeLabelPartition[i];
    }
    for(i=threadIdx.x;i<64;i=i+blockDim.x){
        indexForIndex[256*2+1+1+i] = auxArray[i];
    }
    i = (blockIdx.x * blockDim.x + threadIdx.x)>>5;
    edgeLabelPartition = edgeLabelPartition + intervalNum*2+1+1;
    if(laneId==0) { neigV[warpIdInBlock][32] = 0; }
    __syncthreads();
    totUintNum = indexForIndex[256*2+2];
    uint recordFlag = indexForIndex[256*2+2+1+6+16+16];
    uint writePos = i;
    while (i < partialRowNum){
        if(laneId==0) { neigV[warpIdInBlock][32] = 0; }
        uint predicate,pairIndex,tmp,evidlabel,indexNum=0,svid,evid,presvid,preevid,preevidlabel;
        neigV[warpIdInBlock][laneId] = laneId<embLen?partialEmb[i * embLen + laneId]:1;
        predicate = neigV[warpIdInBlock][laneId]==0;
        predicate = __ballot_sync(0xffffffff,predicate);
        if(predicate>0){ goto zero_nextEmb; }
        pairIndex = 0;
        tmp = indexForIndex[256*2+2+1];
        evidlabel = indexForIndex[256*2+2+1+6];
        svid = (tmp>>5)&0x0000001f;
        evid = tmp&0x0000001f;
        svid = neigV[warpIdInBlock][svid];
        evid = neigV[warpIdInBlock][evid];
        presvid = svid;
        preevid = evid;
        preevidlabel = evidlabel;
        uint lowerLimit, upperLimit, index,baseLen,baseNum,len,useRecord,isRecord,found,lessThan,greatThan;
        uint *baseAddr,*tmpNeigAddr,*recordPos, tmpNeigLen,blockSize;
        recordPos = baseRecordPos+indexForIndex[256*2+2+1+6+16];
        GETBASEADDR
        GETLABEL_ADDR_LEN(0)

        blockSize = tmpNeigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(tmpNeigLen>>5);
        lowerLimit = 0; upperLimit = tmpNeigLen;
        FINDEVID

        for(j=1;j<3;++j){
            pairIndex++;
            evidlabel = indexForIndex[256*2+2+1+6+pairIndex];
            evid = (tmp>>(j*2*5))&0x0000001f;
            svid = (tmp>>((j*2+1)*5))&0x0000001f;
            if(svid==0 || evid==0) { goto normal_nextEmb;}
            svid = neigV[warpIdInBlock][svid];
            evid = neigV[warpIdInBlock][evid];
            if(svid==presvid){
                if(evidlabel==preevidlabel){
                    FINDEVID
                }else{
                    preevidlabel = evidlabel;
                    recordPos = baseRecordPos+indexForIndex[256*2+2+1+6+16+pairIndex];
                    GETLABEL_ADDR_LEN(pairIndex)
                    uint blockSize = tmpNeigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(tmpNeigLen>>5);
                    lowerLimit = 0; upperLimit = tmpNeigLen;
                    FINDEVID
                }
            }else{
                presvid = svid;
                preevidlabel = evidlabel;
                GETBASEADDR
                recordPos = baseRecordPos+indexForIndex[256*2+2+1+6+16+pairIndex];
                GETLABEL_ADDR_LEN(pairIndex)
                uint blockSize = tmpNeigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(tmpNeigLen>>5);
                lowerLimit = 0; upperLimit = tmpNeigLen;
                FINDEVID
            }
        }

        for(j=1;j<totUintNum;++j){
            uint tmp = indexForIndex[256*2+2+1+j];
            for(uint k=0;k<3;++k){
                pairIndex++;
                evidlabel = indexForIndex[256*2+2+1+6+pairIndex];
                evid = (tmp>>(j*2*5))&0x0000001f;
                svid = (tmp>>((j*2+1)*5))&0x0000001f;
                if(svid==0 || evid==0) { goto normal_nextEmb;}
                svid = neigV[warpIdInBlock][svid];
                evid = neigV[warpIdInBlock][evid];
                if(svid==presvid){
                    if(evidlabel==preevidlabel){
                        FINDEVID
                    }else{
                        preevidlabel = evidlabel;
                        recordPos = baseRecordPos+indexForIndex[256*2+2+1+6+16+pairIndex];
                        GETLABEL_ADDR_LEN(pairIndex)
                        uint blockSize = tmpNeigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(tmpNeigLen>>5);
                        lowerLimit = 0; upperLimit = tmpNeigLen;
                        FINDEVID
                    }
                }else{
                    presvid = svid;
                    preevidlabel = evidlabel;
                    GETBASEADDR
                    recordPos = baseRecordPos+indexForIndex[256*2+2+1+6+16+pairIndex];
                    GETLABEL_ADDR_LEN(pairIndex)
                    uint blockSize = tmpNeigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(tmpNeigLen>>5);
                    lowerLimit = 0; upperLimit = tmpNeigLen;
                    FINDEVID
                }
            }
        }
        if(j==totUintNum) { goto normal_nextEmb; }
not_valid_nextEmb:
        if(!isLastPhase){
            if(laneId==0) { partialEmb[i*embLen] = 0; }
        }
zero_nextEmb:
        i = i+gridWarpNum;
        continue;
normal_nextEmb:
        if(!isLastPhase){
            if(writePos<i){
                if(laneId<embLen) { 
                    partialEmb[writePos*embLen+laneId] = neigV[warpIdInBlock][laneId]; 
                    partialEmb[i*embLen+laneId] = 0; 
                }
            }
            writePos = writePos+gridWarpNum;
            if(laneId==0) { neigV[warpIdInBlock][32] = 1; }
        }else{
            if(laneId==0) { writePos = atomicAdd(totWriteRowNum,1); }
            writePos = __shfl_sync(0xffffffff,writePos,0);
            if(laneId<embLen) { partialEmb[writePos*embLen+laneId] = neigV[warpIdInBlock][laneId]; }
        }
        i = i+gridWarpNum;
    }
    if(!isLastPhase){
        if(i>=gridWarpNum) { i=i-gridWarpNum; }
        if(laneId==0 && i<partialRowNum) {
            if(neigV[warpIdInBlock][32]==1){
                writePos = writePos-gridWarpNum;
            }
            writePos = writePos+1;
            atomicMax(totWriteRowNum,writePos); 
        }
    }
}

void reductionPhase(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, uint *neighborsData, uint *baseRecordPos, uint *auxArray, 
    uint *partialEmb, uint intervalNum, uint partialRowNum, uint embLen, bool isLastPhase){

    int numBlocks;
    if(isLastPhase){
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,reductionPhase_kernel<true>,WARPPERBLOCK*32,0);
        reductionPhase_kernel<true><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,totWriteRowNum,edgeLabelPartition,neighborsData,baseRecordPos,auxArray,partialEmb,intervalNum,partialRowNum,embLen);
    }else{
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,reductionPhase_kernel<false>,WARPPERBLOCK*32,0);
        reductionPhase_kernel<false><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,totWriteRowNum,edgeLabelPartition,neighborsData,baseRecordPos,auxArray,partialEmb,intervalNum,partialRowNum,embLen);
    }

}

__global__ void copyPartialEmb_kernel(uint *src, uint *dst, uint numUints){
    uint index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numUints){
        dst[numUints-1-index] = src[numUints-1-index];
    }
}

void copyPartialEmb(uint *src, uint *dst, uint numUints){
    uint blocksize = 8*32;
    uint blockNum = (numUints+blocksize-1)/blocksize;
    copyPartialEmb_kernel<<<blockNum,blocksize>>>(src, dst, numUints);

    /*uint blocksize = 8*32;
    uint blockNum;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockNum,copyPartialEmb_kernel,blocksize,0);
    copyPartialEmb_kernel<<<blockNum,blocksize>>>(src, dst, numUints);*/
}
