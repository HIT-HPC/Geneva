#include "hip/hip_runtime.h"
#include <iostream>
#include "miningPattern.h"
#include "common.h"
//the number of uints
#define INITSHAREDMEMSIZE 128
//#define NBYNMINUSONE (2*256+31)

#define GENEMB_32BY32_1TRANS_RESTRICT(VID,EMBLEN)                                  \
    uint l,tmp1,tmp2,tmp3,tmp4;                                                    \
    uint writeVertex;                                                              \
    for(l=0;l<15*EMBLEN;++l){                                                       \
        uint index = (l<<5)+laneId;                                                \
        tmp1 = index/EMBLEN;                                                       \
        tmp2 = index-tmp1*EMBLEN;                                                  \
        tmp3 = tmp1>>5;                                                            \
        tmp4 = tmp1&31;                                                            \
        if(tmp4<=tmp3){ tmp3 = 30 - tmp3; tmp4 = 31 - tmp4; }                      \
        indexs[0] = __shfl_sync(0xffffffff,VID,tmp3);                              \
        indexs[1] = __shfl_sync(0xffffffff,VID,tmp4);                              \
        writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];              \
        writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];                 \
        newEmb_g[index] = writeVertex;                                             \
    }                                                                              \
    newEmb_g = newEmb_g + 15*32*EMBLEN;\
    for(l=laneId;l<((16*EMBLEN+31)&0xffffffe0);l=l+32){                                                       \
        tmp1 = l/EMBLEN;                                                       \
        tmp2 = l-tmp1*EMBLEN;                                                  \
        indexs[0] = __shfl_sync(0xffffffff,VID,15);                              \
        indexs[1] = __shfl_sync(0xffffffff,VID,tmp1+16);                              \
        writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];              \
        writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];                 \
        if(l<16*EMBLEN) { newEmb_g[l] = writeVertex; }                                             \
    }                                                                              \
    newEmb_g = newEmb_g + 16*EMBLEN;


#define GENEMB_NBYN_1TRANS_RESTRICT(N,VID,EMBLEN)                                  \
    uint l,tmp1,tmp2,tmp3,tmp4,writeVertex;                                        \
    uint tot = ((N*(N-1))>>1)*EMBLEN;                                          \
    uint tot_32 = (tot+31)&0xffffffe0;                                         \
    if((N&1)==1){                                                                  \
        for(uint l=laneId;l<tot_32;l=l+32){                                        \
            tmp1 = l/EMBLEN;                                                       \
            tmp2 = l-tmp1*EMBLEN;                                                  \
            tmp3 = tmp1/N;                                                         \
            tmp4 = tmp1-tmp3*N;                                                    \
            if(tmp3>=tmp4){ tmp3 = N-2-tmp3; tmp4 = N-1-tmp4; }                         \
            indexs[0] = __shfl_sync(0xffffffff,VID,tmp3);                          \
            indexs[1] = __shfl_sync(0xffffffff,VID,tmp4);                          \
            writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];          \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            if(l<tot) { newEmb_g[l] = writeVertex; }                            \
        }                                                                          \
        newEmb_g = newEmb_g + tot;                                                 \
    }else{                                                                         \
        for(uint l=laneId;l<tot_32;l=l+32){                                        \
            tmp1 = l/EMBLEN;                                                       \
            tmp2 = l-tmp1*EMBLEN;                                                  \
            tmp3 = tmp1/(N-1);                                                         \
            tmp4 = tmp1-tmp3*(N-1)+1;                                                    \
            if(tmp4<=tmp3){ tmp3 = N-tmp3-1; tmp4 = N-tmp4; }                    \
            indexs[0] = __shfl_sync(0xffffffff,VID,tmp3);                          \
            indexs[1] = __shfl_sync(0xffffffff,VID,tmp4);                          \
            writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];          \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            if(l<tot) { newEmb_g[l] = writeVertex; }                               \
        }                                                                          \
        newEmb_g = newEmb_g + tot;                                                 \
    }


#define GENEMB_NBYM_1OR2TRANS_NORESTRICT_EVALEQ(OUTERLEN,INNERLEN,OUTERVID,INNERVID,EMBLEN)     \
    uint tot = (OUTERLEN*INNERLEN*EMBLEN);                                                      \
    uint tot_32 = (tot+31)&0xffffffe0;                                                          \
    for(uint l=laneId;l<tot_32;l=l+32){                                                         \
        uint tmp1 = l/EMBLEN;                                                                   \
        uint tmp2 = l-tmp1*EMBLEN;                                                              \
        uint tmp3 = tmp1/INNERLEN;                                                              \
        uint tmp4 = tmp1-tmp3*INNERLEN;                                                         \
        indexs[0] = __shfl_sync(0xffffffff,OUTERVID,tmp3);                                      \
        indexs[1] = __shfl_sync(0xffffffff,INNERVID,tmp4);                                      \
        indexs[0] = indexs[0]==indexs[1]?0:indexs[0];                                           \
        uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];                      \
        writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];                              \
        if(l<tot) { newEmb_g[l] = writeVertex; }                                                \
    }                                                                                           \
    newEmb_g = newEmb_g + tot;


#define GENEMB_NBYM_2TRANS_NOEVALEQ(OUTERLEN,INNERLEN,OUTERVID,INNERVID,EMBLEN)    \
    uint tot = (OUTERLEN*INNERLEN*EMBLEN);                                         \
    uint tot_32 = (tot+31)&0xffffffe0;                                             \
    for(uint l=laneId;l<tot_32;l=l+32){                                            \
        uint tmp1 = l/EMBLEN;                                                      \
        uint tmp2 = l-tmp1*EMBLEN;                                                 \
        uint tmp3 = tmp1/INNERLEN;                                                 \
        uint tmp4 = tmp1-tmp3*INNERLEN;                                            \
        indexs[0] = __shfl_sync(0xffffffff,OUTERVID,tmp3);                         \
        indexs[1] = __shfl_sync(0xffffffff,INNERVID,tmp4);                         \
        uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];         \
        writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];                 \
        if(l<tot) { newEmb_g[l] = writeVertex; }                                   \
    }                                                                              \
    newEmb_g = newEmb_g + tot;


#define GENEMB_NBY32_2TRANS_NOEVALEQ(N,VID1,VID2,EMBLEN)                           \
    for(uint l=0;l<N;++l) {                                                        \
        indexs[0] = __shfl_sync(0xffffffff,VID1,l);                                \
        for(uint m=0;m<EMBLEN;++m) {                                               \
            uint index = (m<<5)+laneId;                                            \
            uint tmp1 = index/EMBLEN;                                              \
            uint tmp2 = index-tmp1*EMBLEN;                                         \
            indexs[1] = __shfl_sync(0xffffffff,VID2,tmp1);                         \
            uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];     \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            newEmb_g[index] = writeVertex;                                         \
        }                                                                          \
        newEmb_g = newEmb_g+32*EMBLEN;                                             \
    }


#define GENEMB_32BYN_2TRANS_NOEVALEQ(N,VID1,VID2,EMBLEN)                           \
    for(uint l=0;l<N;++l) {                                                        \
        indexs[1] = __shfl_sync(0xffffffff,VID2,l);                                \
        for(uint m=0;m<EMBLEN;++m) {                                               \
            uint index = (m<<5)+laneId;                                            \
            uint tmp1 = index/EMBLEN;                                              \
            uint tmp2 = index-tmp1*EMBLEN;                                         \
            indexs[0] = __shfl_sync(0xffffffff,VID1,tmp1);                         \
            uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];     \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            newEmb_g[index] = writeVertex;                                         \
        }                                                                          \
        newEmb_g = newEmb_g+(EMBLEN<<5);                                           \
    }


#define GENEMB_32BY32_2TRANS_NOEVALEQ(VID1,VID2,EMBLEN)                            \
    for(uint l=0;l<32;++l) {                                                       \
        indexs[0] = __shfl_sync(0xffffffff,VID1,l);                                \
        for(uint m=0;m<EMBLEN;++m) {                                               \
            uint index = (m<<5)+laneId;                                            \
            uint tmp1 = index/EMBLEN;                                              \
            uint tmp2 = index-tmp1*EMBLEN;                                         \
            indexs[1] = __shfl_sync(0xffffffff,VID2,tmp1);                         \
            uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];     \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            newEmb_g[index] = writeVertex;                                         \
        }                                                                          \
        newEmb_g = newEmb_g+(EMBLEN<<5);                                           \
    }


#define GENEMB_32BY32_1TRANS_NORESTRICT_EVALEQ(VID,EMBLEN)                         \
    for(uint l=0;l<32;++l) {                                                       \
        indexs[0] = __shfl_sync(0xffffffff,VID,l);                                 \
        for(uint m=0;m<EMBLEN;++m) {                                               \
            uint index = (m<<5)+laneId;                                            \
            uint tmp1 = index/EMBLEN;                                              \
            uint tmp2 = index-tmp1*EMBLEN;                                         \
            indexs[1] = __shfl_sync(0xffffffff,VID,tmp1);                          \
            indexs[1] = indexs[1]==indexs[0]?0:indexs[1];                          \
            uint writeVertex = tmp2<EMBLEN-2?tmpmem_s[tmp2]:indexs[pos1Index];     \
            writeVertex = tmp2<EMBLEN-1?writeVertex:indexs[pos2Index];             \
            newEmb_g[index] = writeVertex;                                         \
        }                                                                          \
        newEmb_g = newEmb_g+(EMBLEN<<5);                                           \
    }


#define REARRANGE(EXTVID,BITFLAG,TOTCOUNT,EQUALS,EQUALE,SHAREDADDR)                           \
    BITFLAG = EXTVID==0?0:1;                                                                   \
    for(uint ii = EQUALS; ii < EQUALE; ++ii) {                                     \
        uint tmp = __shfl_sync(0xffffffff, equalVid, ii);                          \
        if(tmp==EXTVID){ BITFLAG = 0; EXTVID = 0; }                                \
    }                                                                              \
    BITFLAG = __ballot_sync(0xffffffff, BITFLAG );                                 \
    if(BITFLAG==0xffffffff){ TOTCOUNT = 32; }                                      \
    else{                                                                          \
        TOTCOUNT = __popc(BITFLAG);                                                \
        uint mask = 0xffffffff >> (31 - laneId);                                   \
        BITFLAG = mask & BITFLAG;                                                  \
        uint index = __popc(BITFLAG)-1;                                            \
        if(EXTVID>0){ tmpmem_s[SHAREDADDR+index] = EXTVID; }                               \
        EXTVID = tmpmem_s[SHAREDADDR+laneId];                                              \
    }


#define CAL_EQUAL(EXTVID,EQS,EQE)                                                  \
    for(uint ii=EQS;ii<EQE;++ii) {                                                 \
        uint tmp = __shfl_sync(0xffffffff,equalVid,ii);                            \
        EXTVID = tmp==EXTVID?0:EXTVID;                                             \
    }


#define CAL_EQUAL_NUM(EXTVID,COUNT,EQS,EQE)                                        \
    COUNT = 1;                                                                     \
    for(uint ii=EQS;ii<EQE;++ii) {                                                 \
        uint tmp = __shfl_sync(0xffffffff,equalVid,ii);                            \
        if(tmp==EXTVID) { COUNT = 0; EXTVID = 0; }                                 \
    }                                                                              \
    COUNT = __ballot_sync(0xffffffff, COUNT );


#define DETECT_PHASE(EXTVID,EQUALS,EQUALE,PHASEPOS,COUNTINDEX,LABELNUM)              \
    for(uint k=EQUALS;k<EQUALE;++k) {                                              \
        uint tmp = __shfl_sync(0xffffffff,equalVid,k);                             \
        if(tmp==EXTVID) {                                                          \
            uint tmppos = tmpmem_s[COUNTINDEX];                                    \
            tmpmem_s[COUNTINDEX] = tmppos+1;                                      \
            tmpmem_s[32+tmppos+LABELNUM]=PHASEPOS;                                 \
            EXTVID=0;                                                              \
        }                                                                          \
    }

#define GEN_1BY32(INNERVID)                                                         \
    for(uint l=laneId;l<32*embLen;l=l+32) {                                       \
        uint tmp1 = l/embLen;                                                      \
        uint tmp2 = l-tmp1*embLen;                                                 \
        indexs[1] = __shfl_sync(0xffffffff,INNERVID,tmp1);                           \
        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];         \
        writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];                 \
        newEmb_g[l] = writeVertex;                                                 \
    }                                                                              \
    newEmb_g = newEmb_g + 32*embLen;



#define GEN_1BYN(N,INNERVID)                                                        \
    uint tmptot_32 = (N*embLen+31)&0xffffffe0;                                      \
    for(uint l=laneId;l<tmptot_32;l=l+32) {                                         \
        uint tmp1 = l/embLen;                                                       \
        uint tmp2 = l-tmp1*embLen;                                                  \
        indexs[1] = __shfl_sync(0xffffffff,INNERVID,tmp1);                          \
        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];          \
        writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];                  \
        if(l<N*embLen) { newEmb_g[l] = writeVertex; }                               \
    }                                                                               \
    newEmb_g = newEmb_g + N*embLen;


#define ALLOCSPACE(N,EMBLEN)                                                       \
    if(laneId==0){ writePos = atomicAdd(totWriteRowNum,N); }                      \
    writePos = __shfl_sync(0xffffffff,writePos,0);                             \
    newEmb_g = basenewEmb_g+writePos*EMBLEN;                                   



__device__ uint* genInitEmbCore_3V_1and2_noOverlap(uint *neigData1_g, uint *neigData2_g, uint len1, uint len2,
           uint *newEmb_g, uint laneId, uint *tmpmem_s){
    const int embLen = 3;
    uint i,j, len1_32 = len1&0xffffffe0, len2_32 = len2&0xffffffe0, indexs[2];
    if(len1<len2){
        const uint pos1Index = 0, pos2Index = 1;
        for(i=laneId;i<len1_32;i=i+32){
            uint outerVid = neigData1_g[i];
            for(j=laneId;j<len2_32;j=j+32){ uint innerVid = neigData2_g[j]; GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid,innerVid,3) }
            if(len2_32<len2){               uint innerVid = neigData2_g[j]; GENEMB_32BYN_2TRANS_NOEVALEQ((len2-len2_32),outerVid,innerVid,3) }
        }
        if(len1_32<len1){
            uint outerVid = neigData1_g[i];
            for(j=laneId;j<len2_32;j=j+32){ uint innerVid = neigData2_g[j]; GENEMB_NBY32_2TRANS_NOEVALEQ((len1-len1_32),outerVid,innerVid,3) }
            if(len2_32<len2){               uint innerVid = neigData2_g[j]; GENEMB_NBYM_2TRANS_NOEVALEQ((len1-len1_32),(len2-len2_32),outerVid,innerVid,embLen) }
        }
    }else{
        const uint pos1Index = 1, pos2Index = 0;
        for(i=laneId;i<len2_32;i=i+32){
            uint outerVid = neigData2_g[i];
            for(j=laneId;j<len1_32;j=j+32){ uint innerVid = neigData1_g[j]; GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid,innerVid,3) }
            if(len1_32<len1){               uint innerVid = neigData1_g[j]; GENEMB_32BYN_2TRANS_NOEVALEQ((len2-len2_32),outerVid,innerVid,3) }
        }
        if(len2_32<len2){
            uint outerVid = neigData2_g[i];
            for(j=laneId;j<len1_32;j=j+32){ uint innerVid = neigData1_g[j]; GENEMB_NBY32_2TRANS_NOEVALEQ((len1-len1_32),outerVid,innerVid,3) }
            if(len1_32<len1){               
                uint innerVid = neigData1_g[j]; 
                GENEMB_NBYM_2TRANS_NOEVALEQ((len2-len2_32),(len1-len1_32),outerVid,innerVid,embLen)
            }
        }
    }
    return newEmb_g;
}

__device__ uint* genInitEmbCore_3V_1and2_fullOverlap_norestrict(uint *neigData_g, uint len, uint *newEmb_g, uint laneId, uint *tmpmem_s){
    const uint pos1Index = 0, pos2Index = 1;
    uint i,j, len_32 = len&0xffffffe0, indexs[2];
    for(i=laneId;i<len_32;i=i+32) {
        uint outerVid = neigData_g[i];
        GENEMB_32BY32_1TRANS_NORESTRICT_EVALEQ(outerVid,3)
        for(j=laneId;j<i;j=j+32) { uint innerVid = neigData_g[j]; GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid,innerVid,3) }
        for(j=i+32;j<len_32;j=j+32) { uint innerVid = neigData_g[j]; GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid,innerVid,3) }
        if(len_32<len) { uint innerVid = neigData_g[j]; GENEMB_32BYN_2TRANS_NOEVALEQ((len-len_32),outerVid,innerVid,3) }
    }
    if(len_32<len) {
        uint outerVid = neigData_g[i];
        GENEMB_NBYM_1OR2TRANS_NORESTRICT_EVALEQ((len-len_32),(len-len_32),outerVid,outerVid,3)
        for(j=laneId;j<len_32;j=j+32){ uint innerVid = neigData_g[j]; GENEMB_NBY32_2TRANS_NOEVALEQ((len-len_32),outerVid,innerVid,3) }
    }
    return newEmb_g;
}

__device__ uint* genInitEmbCore_3V_1and2_fullOverlap_restrict(uint *neigData_g, uint len, uint *newEmb_g, uint laneId, uint *tmpmem_s){
    uint i,j, len_32 = len&0xffffffe0, indexs[2];

    const uint pos1Index = 1, pos2Index = 0;
    for(i=laneId;i<len_32;i=i+32){
        uint outerVid = neigData_g[i];
        GENEMB_32BY32_1TRANS_RESTRICT(outerVid,3)
        for(j=i+32;j<len_32;j=j+32){ uint innerVid = neigData_g[j]; GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid,innerVid,3) }
        if(len_32<len){ uint innerVid = neigData_g[j]; GENEMB_32BYN_2TRANS_NOEVALEQ((len-len_32),outerVid,innerVid,3) }
    }
    if(len_32<len){ 
        uint outerVid = neigData_g[i]; 
        GENEMB_NBYN_1TRANS_RESTRICT((len-len_32),outerVid,3)
    }
    return newEmb_g;
}

//this is modified
__device__ void genInitEmb_2V(uint *neigData_g,uint len,uint *basenewEmb_g, uint *totWriteRowNum, uint laneId, uint svid,
    uint *tmpmem_s,uint maxRowNum) {

    uint *newEmb_g, writePos;
    ALLOCSPACE(len,2)
    if(writePos+len>=maxRowNum){
        if(laneId==0) {tmpmem_s[0] = maxRowNum;}
        return;
    }
    uint i,j,index=0;
    uint len_32 = len&0xffffffe0;
    uint isodd = laneId&1, fetchIndex = laneId>>1;
    for(i=laneId;i<len_32;i=i+32){
        uint extvid = neigData_g[i];
        uint tmpvid1 = __shfl_sync(0xffffffff,extvid,fetchIndex);
        uint tmpvid2 = __shfl_sync(0xffffffff,extvid,fetchIndex+16);
        uint outerVid = isodd==1?tmpvid1:svid;
        uint innerVid = isodd==1?tmpvid2:svid;
        newEmb_g[index*64+laneId] = outerVid;
        newEmb_g[index*64+32+laneId] = innerVid;
        newEmb_g = newEmb_g + 64;
    }
    if(len_32<len){
        uint extvid = neigData_g[i];
        uint tot = (len-len_32)*2;
        uint tot_32 = (tot+31)&0xffffffe0;
        for(j=laneId;j<tot_32;j=j+32){
            uint tmp1 = j>>1;
            uint tmp2 = j&1;
            uint tmpvid = __shfl_sync(0xffffff,extvid,tmp1);
            uint extvid = tmp2==0?svid:tmpvid;
            if(j<tot){
                newEmb_g[j] = extvid;
            }
        }
    }
    if(laneId==0) { tmpmem_s[0] = 0; }
}

//this is modified
__device__ void genInitEmb_3V_1and2_restric(uint *neigData1_g, uint *neigData2_g, uint len1, uint len2,
           uint *tmpmem_s, uint *basenewEmb_g, uint *totWriteRowNum, uint laneId, uint svid,uint maxRowNum){
    uint *newEmb_g, totWrite=0,writePos;
    if(neigData1_g+len1<=neigData2_g) { if(laneId==0){ tmpmem_s[0] = 0; }return; }
    if(neigData2_g+len2<=neigData1_g){
        ALLOCSPACE((len1*len2),3)
        if(writePos+len1*len2>=maxRowNum){
            if(laneId==0){
                tmpmem_s[0] = maxRowNum;
                tmpmem_s[1] = len1*len2;
                return;
            }
        }
        genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,len1,len2,newEmb_g,laneId,tmpmem_s);
        if(laneId==0){
            tmpmem_s[0]=0;
        }
        return;
    }
    if(neigData1_g < neigData2_g){ len1 = len1 - (neigData2_g-neigData1_g); neigData1_g = neigData2_g; }
    if(neigData1_g+len1<neigData2_g+len2){ len2 = len2-((neigData2_g+len2)-(neigData1_g+len1)); }
    uint tmpn = neigData1_g-neigData2_g;
    uint tmpm = (neigData1_g+len1)-(neigData2_g+len2);
    uint tmph = len1-tmpm;
    totWrite = (tmph*(2*tmpn+tmph-1))/2+tmpm*len2;
    ALLOCSPACE(totWrite,3)
    if(writePos+totWrite>=maxRowNum){
        if(laneId==0){
            tmpmem_s[0] = maxRowNum;
            tmpmem_s[1] = totWrite;
            return;
        }
    }
    if(tmpn>0){
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,len1,tmpn,newEmb_g,laneId,tmpmem_s);
    }
    if(tmpm>0){
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(neigData2_g+len2,neigData2_g,tmpm,len2,newEmb_g,laneId,tmpmem_s);
    }
    genInitEmbCore_3V_1and2_fullOverlap_restrict(neigData1_g,len1-tmpm,newEmb_g,laneId,tmpmem_s);
    if(laneId==0){
        tmpmem_s[0]=0;
    }
}

//this is modified
__device__ void genInitEmb_3V_1and2_noRestrict_sameLabel(uint *neigData1_g, uint *neigData2_g, uint len1, uint len2,
           uint *tmpmem_s, uint *basenewEmb_g, uint *totWriteRowNum, uint laneId,uint maxRowNum) {
    uint *newEmb_g, writePos;
    ALLOCSPACE((len1*len2),3)
    if(writePos+len1*len2>=maxRowNum){
        if(laneId==0){
            tmpmem_s[0] = maxRowNum;
            tmpmem_s[1] = len1*len2;
            return;
        }
    }
    if(neigData1_g+len1<=neigData2_g || neigData2_g+len2<=neigData1_g){
        genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,len1,len2,newEmb_g,laneId,tmpmem_s);
        if(laneId==0){
            tmpmem_s[0]=0;
        }
        return;
    }
    if(neigData1_g<neigData2_g){
        uint tmplen = neigData2_g-neigData1_g;
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,tmplen,len2,newEmb_g,laneId,tmpmem_s);
        neigData1_g = neigData2_g;
        len1 = len1-tmplen;
    }
    if(neigData2_g<neigData1_g){
        uint tmplen = neigData1_g-neigData2_g;
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,len1,tmplen,newEmb_g,laneId,tmpmem_s);
        neigData2_g = neigData1_g;
        len2 = len2-tmplen;
    }
    if(len1<len2){
        uint tmplen = len2-len1;
        len2 = len2 - tmplen;
        uint *tmpaddr = neigData2_g+len2;
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(neigData1_g,tmpaddr,len1,tmplen,newEmb_g,laneId,tmpmem_s);
    }
    if(len2<len1){
        uint tmplen = len1-len2;
        len1 = len1 - tmplen;
        uint *tmpaddr = neigData1_g+len1;
        newEmb_g = genInitEmbCore_3V_1and2_noOverlap(tmpaddr,neigData2_g,tmplen,len2,newEmb_g,laneId,tmpmem_s);
    }
    genInitEmbCore_3V_1and2_fullOverlap_norestrict(neigData1_g,len1,newEmb_g,laneId,tmpmem_s);
    if(laneId==0){
        tmpmem_s[0] = 0;
    }
}

__device__ void genInitEmb_3V_1and2_notsamelabel(uint *neigData1_g, uint *neigData2_g, uint len1, uint len2,
           uint *tmpmem_s, uint *basenewEmb_g, uint *totWriteRowNum, uint laneId, uint maxRowNum) {
    uint *newEmb_g, writePos;
    ALLOCSPACE((len1*len2),3)
    if(writePos+len1*len2>=maxRowNum){
        if(laneId==0) { tmpmem_s[0] = maxRowNum; }
        return;
    }
    genInitEmbCore_3V_1and2_noOverlap(neigData1_g,neigData2_g,len1,len2,newEmb_g,laneId,tmpmem_s);
    if(laneId==0) { tmpmem_s[0] = 0; }
}

__device__ uint* genExtEmbCore_2V_1and2_sameLabel_noRestrict_fulloverlap_detectPhase(uint outerVidnew,uint *neigData_g,uint len,
           uint* tmpmem_s,uint equalVid,uint equalS, uint equalE,uint *newEmb_g,uint laneId,uint embLen) {
    uint j,indexs[2];
    indexs[0] = __shfl_sync(0xffffffff,outerVidnew,0);
    uint len_32 = len&0xffffffe0;
    for(j=laneId;j<len_32;j=j+32) {
        uint innerVidnew = neigData_g[j];
        DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)
        for(uint l=laneId;l<32*embLen;l=l+32) {                                       
            uint tmp1 = l/embLen;                                                    
            uint tmp2 = l-tmp1*embLen;                                               
            indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                       
            uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
            writeVertex = tmp2<embLen-1?writeVertex:indexs[1]; 
            newEmb_g[l] = writeVertex;
            writeVertex = tmp2<embLen-2?writeVertex:indexs[1];
            writeVertex = tmp2<embLen-1?writeVertex:indexs[0]; 
            newEmb_g[l+32*embLen] = writeVertex;
        }                                                                          
        newEmb_g = newEmb_g + 32*embLen*2;
    }
    if(len_32<len){
        uint innerVidnew = laneId<len-len_32?neigData_g[j]:0;
        DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)
        uint tmptot = (len-len_32)*embLen;
        uint tmptot_32 = (tmptot+31)&0xffffffe0;                            
        for(uint l=laneId;l<tmptot_32;l=l+32) {                               
            uint tmp1 = l/embLen;                                             
            uint tmp2 = l-tmp1*embLen;                                        
            indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                
            uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
            writeVertex = tmp2<embLen-1?writeVertex:indexs[1];
            if(l<tmptot) { newEmb_g[l] = writeVertex; }                     
            writeVertex = tmp2<embLen-2?writeVertex:indexs[1];
            writeVertex = tmp2<embLen-1?writeVertex:indexs[0];
            if(l<tmptot) { newEmb_g[l+tmptot] = writeVertex; } 
        }                                                                     
        newEmb_g = newEmb_g + tmptot*2;
    }
    return newEmb_g;
}

template<bool len1great2, bool evaluateEqual>
__device__ uint* genExtEmbCore_2V_1and2_sameLabel_detectPhase(uint outerVidnew,uint *neigData_g,uint len,
           uint* tmpmem_s,uint equalVid,uint equalS, uint equalE,uint *newEmb_g,uint laneId,uint embLen) {
    uint j,indexs[2];
    if(len1great2) { 
        const uint pos1Index = 1, pos2Index = 0;
        indexs[0] = __shfl_sync(0xffffffff,outerVidnew,0);
        uint len_32 = len&0xffffffe0;
        for(j=laneId;j<len_32;j=j+32) {
            uint innerVidnew = neigData_g[j];
            if(evaluateEqual) { innerVidnew = innerVidnew==indexs[0]?0:innerVidnew; }
            DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)
            //GEN_1BY32(innerVid)
            for(uint l=laneId;l<32*embLen;l=l+32) {                                       
                uint tmp1 = l/embLen;                                                    
                uint tmp2 = l-tmp1*embLen;                                               
                indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                       
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];       
                writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index]; 

                newEmb_g[l] = writeVertex;                                              
            }                                                                          
            newEmb_g = newEmb_g + 32*embLen;
        }
        if(len_32<len){
            uint innerVidnew = laneId<len-len_32?neigData_g[j]:0;
            if(evaluateEqual) { innerVidnew = innerVidnew==indexs[0]?0:innerVidnew; }
            DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)
            //GEN_1BYN((len-len_32),innerVid)
            uint tmptot = (len-len_32)*embLen;
            uint tmptot_32 = (tmptot+31)&0xffffffe0;                            
            for(uint l=laneId;l<tmptot_32;l=l+32) {                               
                uint tmp1 = l/embLen;                                             
                uint tmp2 = l-tmp1*embLen;                                        
                indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];
                writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];
     
                if(l<tmptot) { newEmb_g[l] = writeVertex; }                     
            }                                                                     
            newEmb_g = newEmb_g + tmptot;
        }
    }else { 
        const uint pos1Index = 0, pos2Index = 1; 
        indexs[0] = __shfl_sync(0xffffffff,outerVidnew,0);
        uint len_32 = len&0xffffffe0;
        for(j=laneId;j<len_32;j=j+32) {
            uint innerVidnew = neigData_g[j];
            if(evaluateEqual) { innerVidnew = innerVidnew==indexs[0]?0:innerVidnew; }
            DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)
            //GEN_1BY32(innerVid)
            for(uint l=laneId;l<32*embLen;l=l+32) {                                       
                uint tmp1 = l/embLen;                                                    
                uint tmp2 = l-tmp1*embLen;                                               
                indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                       
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];       
                writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index]; 

                newEmb_g[l] = writeVertex;                                              
            }                                                                          
            newEmb_g = newEmb_g + 32*embLen;

        }
        if(len_32<len){
            uint innerVidnew = laneId<len-len_32?neigData_g[j]:0;
            if(evaluateEqual) { innerVidnew = innerVidnew==indexs[0]?0:innerVidnew; }
            DETECT_PHASE(innerVidnew,equalS,equalE,j,32,1)

            //GEN_1BYN((len-len_32),innerVid)
            uint tmptot = (len-len_32)*embLen;
            uint tmptot_32 = (tmptot+31)&0xffffffe0;                                      
            for(uint l=laneId;l<tmptot_32;l=l+32) {                                       
                uint tmp1 = l/embLen;                                                     
                uint tmp2 = l-tmp1*embLen;                                               
                indexs[1] = __shfl_sync(0xffffffff,innerVidnew,tmp1);                       
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];      
                writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];              

                if(l<tmptot) { newEmb_g[l] = writeVertex; }         
            }                                                       
            newEmb_g = newEmb_g + tmptot;
        }
    }
    return newEmb_g;
}

template<bool len1great2, bool evaluateEqual>
__device__ uint* genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase(uint outerVid,uint *neigData_g,uint totcount, uint len,
           uint* tmpmem_s,uint phaseNum,uint *newEmb_g,uint laneId,uint embLen) {
    uint i,j,indexs[2];
    if(len1great2){
        const uint pos1Index = 1, pos2Index = 0;
        uint lowerLimit = 0;
        for(j=0;j<phaseNum;++j) {
            uint upperLimit = tmpmem_s[33+j];
            uint tmpLen = upperLimit-lowerLimit;
            uint tmpLen_32 = tmpLen&0xffffffe0;
            uint k;
            for(k=laneId;k<tmpLen_32;k=k+32) {
                uint innerVid = neigData_g[lowerLimit+k];
                for(i=0;i<totcount;++i) {
                    indexs[0] = __shfl_sync(0xffffffff,outerVid,i);
                    if(evaluateEqual) { innerVid = innerVid==indexs[0]?0:innerVid; }
                    //GEN_1BY32(innerVid)
                    for(uint l=laneId;l<32*embLen;l=l+32) {                                       
                        uint tmp1 = l/embLen;                                                    
                        uint tmp2 = l-tmp1*embLen;                                               
                        indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp1);                      
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];     
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];             

                        newEmb_g[l] = writeVertex;                                             
                    }                                                                          
                    newEmb_g = newEmb_g + 32*embLen;

                }
            }
            if(tmpLen_32<tmpLen) {
                uint innerVid = k<tmpLen?neigData_g[lowerLimit+k]:0;
                if(evaluateEqual) {
                    //GENEMB_NBYM_1OR2TRANS_NORESTRICT_EVALEQ(totcount,(tmpLen-tmpLen_32),outerVid,innerVid,embLen)
                    uint tot = (totcount*(tmpLen-tmpLen_32)*embLen);                                             
                    uint tot_32 = (tot+31)&0xffffffe0;                                                         
                    for(uint l=laneId;l<tot_32;l=l+32){                                                      
                        uint tmp1 = l/embLen;                                                               
                        uint tmp2 = l-tmp1*embLen;                                                          
                        uint tmp3 = tmp1/(tmpLen-tmpLen_32);                                               
                        uint tmp4 = tmp1-tmp3*(tmpLen-tmpLen_32);                                          
                        indexs[0] = __shfl_sync(0xffffffff,outerVid,tmp3);                                
                        indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp4);                                
                        indexs[0] = indexs[0]==indexs[1]?0:indexs[0];                                     
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[pos1Index];                
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[pos2Index];                        
                        if(l<tot) { newEmb_g[l] = writeVertex; }                                               
                    }                                                                                         
                    newEmb_g = newEmb_g + tot;
                }else {
                    GENEMB_NBYM_2TRANS_NOEVALEQ(totcount,(tmpLen-tmpLen_32),outerVid,innerVid,embLen)
                }
            }
            lowerLimit = upperLimit + 1;
        }
    }else { 
        const uint pos1Index = 0, pos2Index = 1; 
        uint lowerLimit = 0;
        for(j=0;j<phaseNum;++j) {
            uint upperLimit = tmpmem_s[33+j];
            uint tmpLen = upperLimit-lowerLimit;
            uint tmpLen_32 = tmpLen&0xffffffe0;
            uint k;
            for(k=laneId;k<tmpLen_32;k=k+32) {
                uint innerVid = neigData_g[lowerLimit+k];
                for(i=0;i<totcount;++i) {
                    indexs[0] = __shfl_sync(0xffffffff,outerVid,i);
                    if(evaluateEqual) { innerVid = innerVid==indexs[0]?0:innerVid; }
                    GEN_1BY32(innerVid)
                }
            }
            if(tmpLen_32<tmpLen) {
                uint innerVid = k<tmpLen?neigData_g[lowerLimit+k]:0;
                if(evaluateEqual) {
                    GENEMB_NBYM_1OR2TRANS_NORESTRICT_EVALEQ(totcount,(tmpLen-tmpLen_32),outerVid,innerVid,embLen)
                }else {
                    GENEMB_NBYM_2TRANS_NOEVALEQ(totcount,(tmpLen-tmpLen_32),outerVid,innerVid,embLen)
                }
            }
            lowerLimit = upperLimit + 1;
        }
    }
    return newEmb_g;
}


//invoked by extemb_1src_2L->genExtEmb_2V_1and2_notSameLabel_withPhase
//tmpmem_s needs to be 96
//this is modified
template<int labelNum, bool isContinue>
__device__ void genExtEmbCore_2V_1and2_noOverlap_withPhase(uint *neigData1_g,uint *neigData2_g,uint len1,uint len2,uint *tmpmem_s,
           uint equalVid, uint equalVNum1,uint equalVNum2, uint *totWriteRowNum, uint *basenewEmb_g, uint laneId, uint embLen,uint maxRowNum) {
    uint equalS1,equalE1,equalS2,equalE2, *newEmb_g,i,totcount,predicate,writePos;
    if(labelNum==1){ equalS1 = 0; equalS2 = 0; equalE1 = equalVNum1; equalE2 = equalVNum2; }
    else {equalS1 = 0; equalE1 = equalVNum1; equalS2 = equalVNum1; equalE2 = equalVNum2; }
    if(len1<len2) {
        if(!isContinue){
            i=laneId; totcount=0;
            uint outerVid, tot_32 = (len1+31)&0xffffffe0;
            while(totcount==0 && i<tot_32) {
                outerVid = i<len1?neigData1_g[i]:0;
                REARRANGE(outerVid,predicate,totcount,equalS1,equalE1,64)
                i=i+32;
            }
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            tmpmem_s[32+laneId] = 0;
            ALLOCSPACE(len2,embLen)
            if(writePos+len2>=maxRowNum){
                if(laneId==0) { tmpmem_s[64] = 1; tmpmem_s[65] = len2; tmpmem_s[66] = maxRowNum; }
                return;
            }
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,false>(outerVid,neigData2_g,len2,tmpmem_s,equalVid,equalS2,equalE2,newEmb_g,laneId,embLen);
            uint phaseNum = tmpmem_s[32];
            totcount = totcount - 1;
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            outerVid = __shfl_down_sync(0xffffffff,outerVid,1);
            ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
            if(tmpmem_s[32+1+phaseNum-1]!=len2-1) {
                if(laneId==0) { tmpmem_s[32+1+phaseNum] = len2; }
                phaseNum = phaseNum+1;
            }
            if(writePos+(len2-phaseNum)*totcount>=maxRowNum){
                if(laneId==0) { tmpmem_s[64] = 2; tmpmem_s[65] = (len2-phaseNum)*totcount; tmpmem_s[66]=maxRowNum; }
                return;
            }
            genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,false>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            if(len1>32) {
                for(;i<tot_32;i=i+32) {
                    outerVid = i<len1?neigData1_g[i]:0;
                    REARRANGE(outerVid,predicate,totcount,equalS1,equalE1,64)
                    if(totcount==0) { continue; }
                    ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
                    if(writePos+(len2-phaseNum)*totcount>=maxRowNum){
                        if(laneId==0) { tmpmem_s[64] = ((i>>5)<<16)|3; tmpmem_s[65] = (len2-phaseNum)*totcount; tmpmem_s[66] = maxRowNum; }
                        return;
                    }
                    genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,false>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
                }
            }
        }else{
            uint retval = tmpmem_s[64];
            uint loopindex = ((retval&0xffff0000)>>16)<<5;
            uint stopindex = retval&0x0000ffff;
            i=laneId; totcount=0;
            uint outerVid, tot_32 = (len1+31)&0xffffffe0;
            while(totcount==0 && i<tot_32) {
                outerVid = i<len1?neigData1_g[i]:0;
                REARRANGE(outerVid,predicate,totcount,equalS1,equalE1,64)
                i=i+32;
            }
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            if(stopindex==1){
                tmpmem_s[32+laneId] = 0;
                ALLOCSPACE(len2,embLen)
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,false>(outerVid,neigData2_g,len2,tmpmem_s,equalVid,equalS2,equalE2,newEmb_g,laneId,embLen);
            }
            uint phaseNum = tmpmem_s[32];
            if(stopindex<=2){
                totcount = totcount - 1;
                if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
                outerVid = __shfl_down_sync(0xffffffff,outerVid,1);
                ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
                if(tmpmem_s[32+1+phaseNum-1]!=len2-1) {
                    if(laneId==0) { tmpmem_s[32+1+phaseNum] = len2; }
                    phaseNum = phaseNum+1;
                }
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,false>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
            if(stopindex<=3){
                if(len1>32) {
                    if(stopindex==3){
                        i = loopindex+laneId;
                    }
                    for(;i<tot_32;i=i+32) {
                        outerVid = i<len1?neigData1_g[i]:0;
                        REARRANGE(outerVid,predicate,totcount,equalS1,equalE1,64)
                        if(totcount==0) { continue; }
                        ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
                        genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,false>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
                    }
                }
            }
        }
    }else {
        if(!isContinue){
            i=laneId; totcount=0;
            uint outerVid, tot_32 = (len2+31)&0xffffffe0;
            while(totcount==0 && i<tot_32) {
                outerVid = i<len2?neigData2_g[i]:0;
                REARRANGE(outerVid,predicate,totcount,equalS2,equalE2,64)
                i=i+32;
            }
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            tmpmem_s[32+laneId] = 0;
            ALLOCSPACE(len1,embLen)
            if(writePos+len1>=maxRowNum){
                if(laneId==0) { tmpmem_s[64] = 1; tmpmem_s[65] = len1; tmpmem_s[66]=maxRowNum; }
                return;
            }
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,false>(outerVid,neigData1_g,len1,tmpmem_s,equalVid,equalS1,equalE1,newEmb_g,laneId,embLen);
            uint phaseNum = tmpmem_s[32];
            totcount = totcount - 1;
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            outerVid = __shfl_down_sync(0xffffffff,outerVid,1);
            ALLOCSPACE((len1-phaseNum)*totcount,embLen)
            if(tmpmem_s[32+1+phaseNum-1]!=len1-1) {
                if(laneId==0) { tmpmem_s[32+1+phaseNum] = len1; }
                phaseNum = phaseNum+1;
                if(laneId==0) { tmpmem_s[32] = phaseNum; }
            }
            if(writePos+(len1-phaseNum)*totcount>=maxRowNum){
                if(laneId==0) { tmpmem_s[64] = 2; tmpmem_s[65] = (len1-phaseNum)*totcount; tmpmem_s[66] = maxRowNum; }
                return;
            }
            genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,false>(outerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            if(len2>32) {
                for(;i<tot_32;i=i+32) {
                    outerVid = i<len2?neigData2_g[i]:0;
                    REARRANGE(outerVid,predicate,totcount,equalS2,equalE2,64)
                    if(totcount==0) { continue; }
                    ALLOCSPACE(((len1-phaseNum)*totcount),embLen)
                    if(writePos+(len1-phaseNum)*totcount>=maxRowNum){
                        if(laneId==0) { tmpmem_s[64] = ((i>>5)<<16)|3; tmpmem_s[65] = (len1-phaseNum)*totcount; tmpmem_s[66]=maxRowNum; }
                        return;
                    }
                    genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,false>(outerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
                }
            }
        }else{
            uint retval = tmpmem_s[64];
            uint loopindex = ((retval&0xffff0000)>>16)<<5;
            uint stopindex = retval&0x0000ffff;
            i=laneId; totcount=0;
            uint outerVid, tot_32 = (len2+31)&0xffffffe0;
            while(totcount==0 && i<tot_32) {
                outerVid = i<len2?neigData2_g[i]:0;
                REARRANGE(outerVid,predicate,totcount,equalS2,equalE2,64)
                i=i+32;
            }
            if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
            if(stopindex==1){
                tmpmem_s[32+laneId] = 0;
                ALLOCSPACE(len1,embLen)
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,false>(outerVid,neigData1_g,len1,tmpmem_s,equalVid,equalS1,equalE1,newEmb_g,laneId,embLen);
            }
            uint phaseNum = tmpmem_s[32];
            if(stopindex<=2){
                totcount = totcount - 1;
                if(totcount==0) { if(laneId==0) { tmpmem_s[66] = 0; } return; }
                outerVid = __shfl_down_sync(0xffffffff,outerVid,1);
                ALLOCSPACE((len1-phaseNum)*totcount,embLen)
                if(tmpmem_s[32+1+phaseNum-1]!=len1-1) {
                    if(laneId==0) { tmpmem_s[32+1+phaseNum] = len1; }
                    phaseNum = phaseNum+1;
                    if(laneId==0) { tmpmem_s[32] = phaseNum; }
                }
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,false>(outerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
            if(stopindex<=3){
                if(len2>32) {
                    if(stopindex==3){
                        i=loopindex+laneId;
                    }
                    for(;i<tot_32;i=i+32) {
                        outerVid = i<len2?neigData2_g[i]:0;
                        REARRANGE(outerVid,predicate,totcount,equalS2,equalE2,64)
                        if(totcount==0) { continue; }
                        ALLOCSPACE(((len1-phaseNum)*totcount),embLen)
                        genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,false>(outerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
                    }
                }
            }
        }
    }
}

//this need to be modified according to
//genExtEmbCore_2V_1and2_noRestrict_fullOverlap_withPhase
//this is modified
template<uint pos1Index, uint pos2Index, bool isContinue>
__device__ void genExtEmbCore_2V_1and2_restrict_fullOverlap_withPhase(uint *neigData_g,uint len,uint *tmpmem_s, uint equalVid,
           uint equalVNum, uint *totWriteRowNum, uint *basenewEmb_g, uint laneId, uint embLen, uint maxRowNum) {
    uint len_32 = len&0xffffffe0,*newEmb_g, tot_32 = (len+31)&0xffffffe0, indexs[2];
    uint i,j,predicate,totcount,outerVid,outerLowerLimit,writePos;

    i=laneId; totcount = 0;
    uint skipNum = 0;
    uint retval = tmpmem_s[64];
    while(totcount==0 && i<tot_32) {
        outerVid = i<len?neigData_g[i]:0;
        predicate = outerVid==0?0:1;                                                               
        for(uint ii = 0; ii < equalVNum; ++ii) {                                     
            uint tmp = __shfl_sync(0xffffffff, equalVid, ii);                        
            if(tmp==outerVid){ predicate = 0; outerVid = 0; }                        
        }                                                                           
        predicate = __ballot_sync(0xffffffff, predicate); 
        if(predicate==0xffffffff){ 
            totcount = 32; 
            break;
        }
        totcount = __popc(predicate);                                            
        if(totcount==0){
            i=i+32;
            continue;
        }
        skipNum = (i&0xffffffe0)+__ffs(predicate)-1;
        uint mask = 0xffffffff >> (31 - laneId);                               
        predicate = mask & predicate;                                              
        uint index = __popc(predicate)-1;                                        
        if(outerVid>0){ tmpmem_s[64+index] = outerVid; }                   
        outerVid = tmpmem_s[64+laneId];
        break;                  
    }
    if(totcount==0){ if(laneId==0){ tmpmem_s[66]=0; }return; }
    neigData_g = neigData_g+skipNum+1;
    len = len-skipNum-1;
    uint phaseNum;
    if(!isContinue){
        tmpmem_s[32+laneId] = 0;
        ALLOCSPACE(len,embLen)
        if(writePos+len>=maxRowNum){
            if(laneId==0) { tmpmem_s[64] = 1; tmpmem_s[65] = len; tmpmem_s[66] = maxRowNum; }
            return;
        }
        if(pos1Index>pos2Index){
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,false>(outerVid, neigData_g, len,tmpmem_s, equalVid, 0, equalVNum, newEmb_g,laneId, embLen);
        }else{
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,false>(outerVid, neigData_g, len,tmpmem_s, equalVid, 0, equalVNum, newEmb_g,laneId, embLen);
        }
        phaseNum = tmpmem_s[32];
        ALLOCSPACE((((len-phaseNum)*(len-phaseNum-1))>>1),embLen)
        if(writePos+(((len-phaseNum)*(len-phaseNum-1))>>1)>=maxRowNum){
            if(laneId==0) { tmpmem_s[64] = 2; tmpmem_s[65] = (((len-phaseNum)*(len-phaseNum-1))>>1); tmpmem_s[66] = maxRowNum; }
            return;
        }
        if(tmpmem_s[32+1+phaseNum-1]!=len-1) {
            if(laneId==0) { tmpmem_s[32+1+phaseNum] = len; }
            phaseNum = phaseNum+1;
        }
    }else{
        uint stopindex = retval & 0x0000000f;
        if(stopindex==1){
            tmpmem_s[32+laneId] = 0;
            ALLOCSPACE(len,embLen)
            if(pos1Index>pos2Index){
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,false>(outerVid, neigData_g, len,tmpmem_s, equalVid, 0, equalVNum, newEmb_g,laneId, embLen);
            }else{
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,false>(outerVid, neigData_g, len,tmpmem_s, equalVid, 0, equalVNum, newEmb_g,laneId, embLen);
            }
        }
        if(stopindex<=2){
            phaseNum = tmpmem_s[32];
            ALLOCSPACE((((len-phaseNum)*(len-phaseNum-1))>>1),embLen)
            if(tmpmem_s[32+1+phaseNum-1]!=len-1) {
                if(laneId==0) { tmpmem_s[32+1+phaseNum] = len; }
                phaseNum = phaseNum+1;
            }
        }
    }
    outerLowerLimit = 0;
    for(i=0;i<phaseNum;++i) {
        uint outerUpperLimit = tmpmem_s[33+i];
        uint outerLen = outerUpperLimit-outerLowerLimit;
        uint outerLen_32 = outerLen&0xffffffe0;
        for(j=laneId;j<outerLen_32;j=j+32) {
            outerVid = neigData_g[outerLowerLimit+j];
            GENEMB_32BY32_1TRANS_RESTRICT(outerVid,embLen)
            uint k;
            for(k=j+32;k<outerLen_32;k=k+32) {
                uint innerVid = neigData_g[outerLowerLimit+k];
                GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid, innerVid, embLen)
            }
            if(outerLen_32<outerLen) {
                uint innerVid = neigData_g[outerLowerLimit+k];
                GENEMB_32BYN_2TRANS_NOEVALEQ((outerLen - outerLen_32), outerVid, innerVid, embLen)
            }
            uint innerLowerLimit = outerUpperLimit+1, innerUpperLimit;
            for(k=i+1;k<phaseNum;++k) {
                innerUpperLimit = tmpmem_s[33+k];
                uint innerLen = innerUpperLimit-innerLowerLimit;
                uint innerLen_32 = innerLen&0xffffffe0;
                uint g;
                for(g=laneId;g<innerLen_32;g=g+32) {
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    GENEMB_32BY32_2TRANS_NOEVALEQ(outerVid, innerVid, embLen)
                }
                if(innerLen_32<innerLen) {
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    GENEMB_32BYN_2TRANS_NOEVALEQ((innerLen - innerLen_32), outerVid, innerVid, embLen)
                }
                innerLowerLimit = innerUpperLimit + 1;
            }
        }
        if(outerLen_32<outerLen) {
            outerVid = neigData_g[outerLowerLimit+j];
            GENEMB_NBYN_1TRANS_RESTRICT((outerLen - outerLen_32), outerVid, embLen)
            uint innerLowerLimit = outerUpperLimit+1, innerUpperLimit;
            uint k;
            for(k=i+1;k<phaseNum;++k) {
                innerUpperLimit = tmpmem_s[33+k];
                uint innerLen = innerUpperLimit-innerLowerLimit;
                uint innerLen_32 = innerLen&0xffffffe0;
                uint g;
                for(g=laneId;j<innerLen_32;j=j+32) {
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    GENEMB_NBY32_2TRANS_NOEVALEQ((outerLen - outerLen_32), outerVid, innerVid, embLen)
                }
                if(innerLen_32<innerLen) {
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    GENEMB_NBYM_2TRANS_NOEVALEQ((outerLen-outerLen_32),(innerLen-innerLen_32),outerVid,innerVid,embLen)
                }
                innerLowerLimit = innerUpperLimit+1;
            }
        }
        outerLowerLimit = outerUpperLimit+1;
    }
    if(laneId==0){
        tmpmem_s[66] = 0;
    }
}

//this is modified
template<bool isContinue>
__device__ void genExtEmbCore_2V_1and2_noRestrict_fullOverlap_withPhase(uint *neigData_g,uint len,uint *tmpmem_s, uint equalVid,
           uint equalS,uint equalE, uint *totWriteRowNum, uint *basenewEmb_g, uint laneId, uint embLen,uint maxRowNum) {
    uint *newEmb_g, indexs[2];
    uint i,j,predicate, totcount,outerVid,outerLowerLimit,tot_32,writePos;
    const uint pos1Index = 0, pos2Index = 1;
    i=laneId; totcount = 0, tot_32 = (len+31)&0xffffffffe0;
    uint skipNum=0;
    uint retval = tmpmem_s[64];
    while(totcount==0 && i<tot_32) {
        outerVid = i<len?neigData_g[i]:0;
        predicate = outerVid==0?0:1;                                                         
        for(uint ii = equalS; ii < equalE; ++ii) {                                     
            uint tmp = __shfl_sync(0xffffffff, equalVid, ii);                        
            if(tmp==outerVid){ predicate = 0; outerVid = 0; }                        
        }                                                                           
        predicate = __ballot_sync(0xffffffff, predicate); 
        if(predicate==0xffffffff){ 
            totcount = 32; 
            break;
        }
        totcount = __popc(predicate);                                            
        if(totcount==0){
            i=i+32;
            continue;
        }
        skipNum = (i&0xffffffe0)+__ffs(predicate)-1;
        uint mask = 0xffffffff >> (31 - laneId);                               
        predicate = mask & predicate;                                              
        uint index = __popc(predicate)-1;                                        
        if(outerVid>0){ tmpmem_s[64+index] = outerVid; }                   
        outerVid = tmpmem_s[64+laneId];
        break;                  
    }
    if(totcount==0){ if(laneId==0) { tmpmem_s[66]=0; } return; }
    neigData_g = neigData_g + skipNum+1;
    len = len-skipNum-1;
    if(len==0) { if(laneId==0) { tmpmem_s[66]=0; } return; }
    uint phaseNum;
    if(!isContinue){
        tmpmem_s[32+laneId] = 0;
        ALLOCSPACE((len*2),embLen)
        if(writePos+len*2>=maxRowNum){
            if(laneId==0) { tmpmem_s[64]=1; tmpmem_s[65]=len*2; tmpmem_s[66]=maxRowNum; }
            return;
        }
        genExtEmbCore_2V_1and2_sameLabel_noRestrict_fulloverlap_detectPhase(outerVid,neigData_g,len,tmpmem_s,equalVid,equalS,equalE,newEmb_g,laneId,embLen);
        phaseNum = tmpmem_s[32];
        if(len-phaseNum-1==0) { if(laneId==0) { tmpmem_s[66]=0; } return; }
        ALLOCSPACE(((len-phaseNum-1)*(len-phaseNum)),embLen)
        if(writePos+((len-phaseNum-1)*(len-phaseNum))>=maxRowNum){
            if(laneId==0) { tmpmem_s[64]=2; tmpmem_s[65]=((len-phaseNum-1)*(len-phaseNum)); tmpmem_s[66]=maxRowNum; }
            return;
        }
        if(tmpmem_s[32+1+phaseNum-1]!=len-1) {
            if(laneId==0) { tmpmem_s[32+1+phaseNum] = len; }
            phaseNum = phaseNum+1;
        }
    }else{
        uint stopindex = retval&0x0000000f;
        if(stopindex==1){
            tmpmem_s[32+laneId] = 0;
            ALLOCSPACE((len*2),embLen)
            genExtEmbCore_2V_1and2_sameLabel_noRestrict_fulloverlap_detectPhase(outerVid,neigData_g,len,tmpmem_s,equalVid,equalS,equalE,newEmb_g,laneId,embLen);
        }
        if(stopindex<=2){
            phaseNum = tmpmem_s[32];
            ALLOCSPACE(((len-phaseNum-1)*(len-phaseNum)),embLen)
            if(tmpmem_s[32+1+phaseNum-1]!=len-1) {
                if(laneId==0) { tmpmem_s[32+1+phaseNum] = len; }
                phaseNum = phaseNum+1;
            }
        }
    }
    outerLowerLimit = 0;
    for(i=0;i<phaseNum;++i) {
        uint outerUpperLimit = tmpmem_s[33+i];
        uint outerLen = outerUpperLimit-outerLowerLimit;
        uint outerLen_32 = outerLen&0xffffffe0;
        for(j=laneId;j<outerLen_32;j=j+32) {
            outerVid = neigData_g[outerLowerLimit+j];
            for(uint l=laneId;l<32*31*embLen;l=l+32){
                uint tmp1 = l/embLen;
                uint tmp2 = l-tmp1*embLen;
                uint col = (tmp1>>5);
                uint row = tmp1-col*32;
                col = row<=col?col+1:col;
                row = __shfl_sync(0xffffffff,outerVid,row);                       
                col = __shfl_sync(0xffffffff,outerVid,col);
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                writeVertex = tmp2<embLen-1?writeVertex:col;
                newEmb_g[l] = writeVertex;
            }
            newEmb_g = newEmb_g + 32*31*embLen;
            for(uint k=j+32;k<outerLen_32;k=k+32){
                uint innerVid = neigData_g[outerLowerLimit+k];
                for(uint l=laneId;l<32*32*embLen;l=l+32){
                    uint tmp1 = l/embLen;
                    uint tmp2 = l-tmp1*embLen;
                    uint row = tmp1>>5;
                    uint col = tmp1-row*32;
                    row = __shfl_sync(0xffffffff,outerVid,row);                       
                    col = __shfl_sync(0xffffffff,innerVid,col);
                    uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                    writeVertex = tmp2<embLen-1?writeVertex:col;
                    newEmb_g[l] = writeVertex;
                    writeVertex = tmp2<embLen-2?writeVertex:col;
                    writeVertex = tmp2<embLen-1?writeVertex:row;
                    newEmb_g[32*32*embLen+l] = writeVertex;
                }
                newEmb_g = newEmb_g + 32*32*embLen*2;
            }
            if(outerLen_32<outerLen){
                uint innerVid = laneId<outerLen-outerLen_32?neigData_g[outerLowerLimit+outerLen_32+laneId]:0;
                uint tmptot = 32*(outerLen-outerLen_32)*embLen;
                for(uint l=laneId;l<tmptot;l=l+32){
                    uint tmp1 = l/embLen;
                    uint tmp2 = l-tmp1*embLen;
                    uint row = tmp1>>5;
                    uint col = tmp1-row*32;
                    row = __shfl_sync(0xffffffff,innerVid,row);                       
                    col = __shfl_sync(0xffffffff,outerVid,col);
                    uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                    writeVertex = tmp2<embLen-1?writeVertex:col;
                    newEmb_g[l] = writeVertex;
                    writeVertex = tmp2<embLen-2?writeVertex:col;
                    writeVertex = tmp2<embLen-1?writeVertex:row;
                    newEmb_g[tmptot+l] = writeVertex;
                }
                newEmb_g = newEmb_g + tmptot*2;
            }
            uint k, innerLowerLimit, innerUpperLimit;
            innerLowerLimit = outerUpperLimit+1;
            for(k=i+1;k<phaseNum;++k) {
                innerUpperLimit = tmpmem_s[33+k];
                uint innerLen = innerUpperLimit-innerLowerLimit;
                uint innerLen_32 = innerLen&0xffffffe0;
                for(uint g=laneId;g<innerLen_32;g=g+32){ 
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    for(uint l=laneId;l<32*32*embLen;l=l+32){
                        uint tmp1 = l/embLen;
                        uint tmp2 = l-tmp1*embLen;
                        uint row = tmp1>>5;
                        uint col = tmp1-row*32;
                        row = __shfl_sync(0xffffffff,outerVid,row);                       
                        col = __shfl_sync(0xffffffff,innerVid,col);
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                        writeVertex = tmp2<embLen-1?writeVertex:col;
                        newEmb_g[l] = writeVertex;
                        writeVertex = tmp2<embLen-2?writeVertex:col;
                        writeVertex = tmp2<embLen-1?writeVertex:row;
                        newEmb_g[32*32*embLen+l] = writeVertex;
                    }
                    newEmb_g = newEmb_g + 32*32*embLen*2;
                }
                if(innerLen_32<innerLen) { 
                    uint innerVid = laneId<innerLen-innerLen_32?neigData_g[innerLowerLimit+innerLen_32+laneId]:0; 
                    uint tmptot = 32*(innerLen-innerLen_32)*embLen;
                    for(uint l=laneId;l<tmptot;l=l+32){
                        uint tmp1 = l/embLen;
                        uint tmp2 = l-tmp1*embLen;
                        uint row = tmp1>>5;
                        uint col = tmp1-row*32;
                        row = __shfl_sync(0xffffffff,innerVid,row);                       
                        col = __shfl_sync(0xffffffff,outerVid,col);
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                        writeVertex = tmp2<embLen-1?writeVertex:col;
                        newEmb_g[l] = writeVertex;
                        writeVertex = tmp2<embLen-2?writeVertex:col;
                        writeVertex = tmp2<embLen-1?writeVertex:row;
                        newEmb_g[tmptot+l] = writeVertex;
                    }
                    newEmb_g = newEmb_g + tmptot*2;
                }
                innerLowerLimit = innerUpperLimit + 1;
            }
        }
        if(outerLen_32<outerLen) {
            outerVid = laneId<outerLen-outerLen_32?neigData_g[outerLowerLimit+outerLen_32+laneId]:0;
            uint tmptot = (outerLen-outerLen_32)*(outerLen-outerLen_32-1)*embLen;
            uint tmptot_32 = (tmptot+31)&0xffffffe0;
            for(uint l=laneId;l<tmptot_32;l=l+32){
                uint tmp1 = l/embLen;
                uint tmp2 = l-tmp1*embLen;
                uint col = tmp1/(outerLen-outerLen_32);
                uint row = tmp1-col*(outerLen-outerLen_32);
                col = row<=col?col+1:col;
                row = __shfl_sync(0xffffffff,outerVid,row);                       
                col = __shfl_sync(0xffffffff,outerVid,col);
                uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                writeVertex = tmp2<embLen-1?writeVertex:col;
                if(l<tmptot) { newEmb_g[l] = writeVertex; }
            }
            newEmb_g = newEmb_g + tmptot;   
            uint k,innerLowerLimit = outerUpperLimit+1,innerUpperLimit;
            for(k=i+1;k<phaseNum;++k) {
                innerUpperLimit = tmpmem_s[33+k];
                uint innerLen = innerUpperLimit-innerLowerLimit;
                uint innerLen_32 = innerLen&0xffffffe0;
                for(uint g=laneId;g<innerLen_32;g=g+32) { 
                    uint innerVid = neigData_g[innerLowerLimit+g];
                    uint tmptot = 32*(outerLen-outerLen_32)*embLen;
                    for(uint l=laneId;l<tmptot;l=l+32){
                        uint tmp1 = l/embLen;
                        uint tmp2 = l-tmp1*embLen;
                        uint row = tmp1>>5;
                        uint col = tmp1-row*32;
                        row = __shfl_sync(0xffffffff,outerVid,row);                       
                        col = __shfl_sync(0xffffffff,innerVid,col);
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                        writeVertex = tmp2<embLen-1?writeVertex:col;
                        newEmb_g[l] = writeVertex;
                        writeVertex = tmp2<embLen-2?writeVertex:col;
                        writeVertex = tmp2<embLen-1?writeVertex:row;
                        newEmb_g[tmptot+l] = writeVertex;
                    }
                    newEmb_g = newEmb_g + tmptot*2;
                }
                if(innerLen_32<innerLen) { 
                    uint innerVid = laneId<innerLen-innerLen_32?neigData_g[innerLowerLimit+innerLen_32+laneId]:0;
                    uint tmptot = (outerLen-outerLen_32)*(innerLen-innerLen_32)*embLen;
                    uint tmptot_32 = (tmptot+31)&0xffffffe0;
                    for(uint l=laneId;l<tmptot_32;l=l+32){
                        uint tmp1=l/embLen;
                        uint tmp2 = l-tmp1*embLen;
                        uint row = tmp1/(outerLen-outerLen_32);
                        uint col = tmp1-row*(outerLen-outerLen_32);
                        row = __shfl_sync(0xffffffff,innerVid,row);                       
                        col = __shfl_sync(0xffffffff,outerVid,col);
                        uint writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:row;
                        writeVertex = tmp2<embLen-1?writeVertex:col;
                        newEmb_g[l] = writeVertex;
                        writeVertex = tmp2<embLen-2?writeVertex:col;
                        writeVertex = tmp2<embLen-1?writeVertex:row;
                        newEmb_g[tmptot+l] = writeVertex;
                    }
                    newEmb_g = newEmb_g + tmptot*2;
                }
                innerLowerLimit = innerUpperLimit + 1;
            }
        }
        outerLowerLimit = outerUpperLimit+1;
    }
    if(laneId==0){ tmpmem_s[66] = 0; }
}

template<bool isLastPhase, bool isContinue>
__device__ void genExtEmb_1V(uint *neigData_g, uint len, uint *tmpmem_s, uint *equalVertices_s,
    uint *totWriteRowNum, uint *basenewEmb_g, uint laneId, uint embLen, uint maxRowNum) {

    uint *newEmb_g,i,writePos;
    uint equalVNum = equalVertices_s[0];
    uint equalVPos = equalVertices_s[2+laneId];
    uint equalVid = laneId<equalVNum?tmpmem_s[equalVPos]:0;

    if(!isLastPhase){
        ALLOCSPACE(len,embLen)
        if(writePos+len>=maxRowNum){
            if(laneId==0) { tmpmem_s[32] = 0xffff0000; tmpmem_s[33] = len; tmpmem_s[34] = maxRowNum; }
            return;
        }
        uint len_32 = len&0xffffffe0;
        for(i=laneId;i<len_32;i=i+32) {
            uint extvid = neigData_g[i];
            CAL_EQUAL(extvid,0,equalVNum)
            for(uint j=laneId;j<32*embLen;j=j+32) {
                uint tmp1 = j/embLen;
                uint tmp2 = j-tmp1*embLen;
                uint tmpvid = __shfl_sync(0xffffffff,extvid,tmp1);
                uint writeVertex = tmp2<embLen-1?tmpmem_s[tmp2]:tmpvid;
                newEmb_g[j] = writeVertex;
            }
            newEmb_g = newEmb_g+32*embLen;
        }
        if(len_32<len) {
            uint extvid = neigData_g[i];
            CAL_EQUAL(extvid,0,equalVNum)
            uint tmp = (len-len_32)*embLen;
            uint tmp_32 = (tmp+31)&0xffffffe0;
            for(uint j=laneId;j<tmp_32;j=j+32) {
                uint tmp1 = j/embLen;
                uint tmp2 = j-tmp1*embLen;
                uint tmpvid = __shfl_sync(0xffffffff,extvid,tmp1);
                uint writeVertex = tmp2<embLen-1?tmpmem_s[tmp2]:tmpvid;
                if(j<tmp) { newEmb_g[j] = writeVertex; }
            }
        }
    }else{
        uint len_32 = (len+31)&0xffffffe0, predicate,totcount;
        if(isContinue){
            uint retval = tmpmem_s[3];
            i=(retval>>16)+laneId;
        }else{
            i=laneId;
        }
        while(i<len_32) {
            totcount=0;
            uint extvid = i<len?neigData_g[i]:0;
            REARRANGE(extvid,predicate,totcount,0,equalVNum,32)
            ALLOCSPACE(totcount,embLen)
            if(writePos+totcount>=maxRowNum){
                if(laneId==0) { tmpmem_s[32] = ((i>>5)<<16)|0x00000001; tmpmem_s[33] = totcount; tmpmem_s[34] = maxRowNum;}
                return;
            }
            uint tmp = totcount*embLen;
            uint tmp_32 = (tmp+31)&0xffffffe0;
            for(uint j=laneId;j<tmp_32;j=j+32) {
                uint tmp1 = j/embLen;
                uint tmp2 = j-tmp1*embLen;
                uint tmpvid = __shfl_sync(0xffffffff,extvid,tmp1);
                uint writeVertex = tmp2<embLen-1?tmpmem_s[tmp2]:tmpvid;
                if(j<tmp) { newEmb_g[j] = writeVertex; }
            }
            newEmb_g = newEmb_g+tmp;
            i=i + 32;
        }
    }
    if(laneId==0){ tmpmem_s[34]=0; }
}

//invoked by extEmb_2V_1src_1and2_sameLabel_NoHash_kernel
//tmpmem_s 96
//this is modified
template<bool isContinue>
__device__ void genExtEmb_2V_1src_1and2_restrict_withPhase(uint *neigData1_g, uint *neigData2_g,
           uint len1, uint len2, uint *tmpmem_s, uint *equalVertices_s, uint *totWriteRowNum, 
           uint *basenewEmb_g, uint laneId, uint embLen, uint maxRowNum) {

    uint i, predicate,tmp;
    uint equalVNum = equalVertices_s[1];
    uint equalVPos = equalVertices_s[2+laneId];
    uint equalVid = laneId<equalVNum?tmpmem_s[equalVPos]:0;
    if(isContinue) { tmp = tmpmem_s[32+laneId]; }
    for(i=0;i<equalVNum;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = equalVPos>equalVid;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate)-(32-equalVNum);
        if(laneId==0){
            tmpmem_s[32+predicate] = equalVPos;
        }
    }
    equalVid = laneId<equalVNum?tmpmem_s[32+laneId]:0;
    if(isContinue) { tmpmem_s[32+laneId] = tmp; }
    if(neigData1_g+len1<=neigData2_g) { return; }
    if(neigData2_g+len2<=neigData1_g) {
        genExtEmbCore_2V_1and2_noOverlap_withPhase<1,isContinue>(neigData1_g,neigData2_g,len1,len2,tmpmem_s,equalVid,equalVNum,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
        return;
    }
    if(!isContinue){
        if(neigData1_g<neigData2_g) { neigData1_g = neigData2_g; len1 = len1- (neigData2_g-neigData1_g); }
        else if(neigData1_g>neigData2_g){
            uint tmpLen = neigData1_g-neigData2_g;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData1_g,neigData2_g,len1,tmpLen,tmpmem_s,equalVid,equalVNum,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
            if(tmpmem_s[66]==maxRowNum){
                tmp = tmpmem_s[64];
                tmp = (tmp<<4) | 0x00000001;
                tmp = tmp&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000) | tmp; }
                return;
            }
            neigData2_g = neigData1_g;
            len2 = len2 - tmpLen;
        }
        if(len1<len2) { len2 = len1; }
        else if(len1>len2){
            uint tmpLen = len1-len2;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData2_g+len2,neigData2_g,tmpLen,len2,tmpmem_s,equalVid,equalVNum,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
            if(tmpmem_s[66]==maxRowNum){
                tmp = tmpmem_s[64];
                tmp = (tmp<<4) | 0x00000002;
                tmp = tmp&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000) | tmp; }
                return;
            }
            len1 = len1-tmpLen;
        }
        genExtEmbCore_2V_1and2_restrict_fullOverlap_withPhase<1,0,false>(neigData1_g,len1,tmpmem_s,equalVid,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
        if(tmpmem_s[66]==maxRowNum){
            tmp = tmpmem_s[64];
            tmp = (tmp<<4) | 0x00000003;
            tmp = tmp&0x0000ffff;
            if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000) | tmp; }
            return;
        }
    }else{
        uint tmp = tmpmem_s[64+5];
        uint stopindex = tmp&0x0000000f;
        tmpmem_s[64+5] = ((tmp&0x0000ffff)>>4)|(tmp&0xffff0000);
        if(neigData1_g<neigData2_g) { neigData1_g = neigData2_g; len1 = len1- (neigData2_g-neigData1_g); }
        else if(neigData1_g>neigData2_g){
            uint tmpLen = neigData1_g-neigData2_g;
            if(stopindex==1){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData1_g,neigData2_g,len1,tmpLen,tmpmem_s,equalVid,equalVNum,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
            }
            neigData2_g = neigData1_g;
            len2 = len2 - tmpLen;
        }
        if(len1<len2) { len2 = len1; }
        else if(len1>len2){
            uint tmpLen = len1-len2;
            if(stopindex<=2){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData2_g+len2,neigData2_g,tmpLen,len2,tmpmem_s,equalVid,equalVNum,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
            }
            len1 = len1-tmpLen;
        }
        genExtEmbCore_2V_1and2_restrict_fullOverlap_withPhase<1,0,true>(neigData1_g,len1,tmpmem_s,equalVid,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
    }
}

//invoked by extEmb_2V_2src_1L, this uses 96 shared memory
//this is modified
template<bool len1SmallLen2,bool isContinue>
__device__ void genExtEmb_2V_2src_1and2_restrict(uint *neigData1_g, uint *neigData2_g,uint len1, uint len2, uint *tmpmem_s,
           uint *equalVertices_s, uint *totWriteRowNum, uint *newEmb_g,uint *basenewEmb_g, uint laneId, uint embLen,uint maxRowNum) {
    
    uint i,j,indexs[2],outerLen, innerLen, outerVid,predicate,writePos;
    uint equalVNum = equalVertices_s[0];
    uint equalVPos = equalVertices_s[1+laneId];
    uint equalVid = laneId<equalVNum?tmpmem_s[equalVPos]:0;
    for(i=0;i<equalVNum;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = equalVPos>equalVid;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate)-(32-equalVNum);
        if(laneId==0){
            tmpmem_s[64+predicate] = equalVPos;
        }
    }
    equalVid = laneId<equalVNum?tmpmem_s[64+laneId]:0;
    //this is used to store rearranged outer vertices
    if(len1SmallLen2){ outerLen = len1; innerLen = len2; }
    else { outerLen = len2; innerLen = len1; }
    bool isfirst;
    if(isContinue){
        isfirst=true;
        i = tmpmem_s[32+5]+laneId;
    }else{
        i = laneId;
        tmpmem_s[32+laneId] = 0;
    }
    for(;i<((outerLen+31)&0xffffffe0);i=i+32) {
        if (len1SmallLen2) { outerVid = i < outerLen ? neigData1_g[outerLen - 1 - i] : 0; }
        else { outerVid = i < outerLen ? neigData2_g[i] : 0; }
        uint predicate,totcount;
        REARRANGE(outerVid,predicate,totcount,0,equalVNum,64)
        if(totcount == 0) { continue; }
        int boundryOfNoNeedEval = -1,newInnerLen = 0;
        bool notOverMaxRowNum=true;
        if(isContinue){
            if(isfirst){
                isfirst=false;
                outerVid = __shfl_down_sync(0xffffffff,outerVid,tmpmem_s[32+8]);
                totcount = totcount-tmpmem_s[32+8];
                j = tmpmem_s[32+6]+laneId;
                innerLen = tmpmem_s[32+7]; 
            }else{
                j=laneId;
            }
        }else{
            j = laneId;
        }
        for(;j<((innerLen+31)&0xffffffe0);j=j+32) {
            uint innerVid;
            if (len1SmallLen2) { innerVid = j<innerLen?neigData2_g[innerLen-1-j]:0; }
            else { innerVid = j<innerLen?neigData1_g[j]:0; }
            for(uint k=0;k<equalVNum;++k) { 
                uint tmp=__shfl_sync(0xffffffff,equalVid,k);
                innerVid = tmp==innerVid?0:innerVid; 
            }
            uint transsize = j<(innerLen&0xffffffe0)?32:innerLen-(innerLen&0xffffffe0);
            for(int k=0;k<=boundryOfNoNeedEval;++k) {
                indexs[0] = __shfl_sync(0xffffffff,outerVid,k);
                //tmpmem_s[64+k] is the distance between basenewEmb_g and the current pos of writed uints
                uint tmp = tmpmem_s[64+k];
                newEmb_g = basenewEmb_g + tmp;
                uint len = transsize*embLen;
                uint len_32 = len&0xffffffe0;
                uint l;
                for(l=laneId;l<len_32;l=l+32) {
                    uint tmp1 = l/embLen;
                    uint tmp2 = l-tmp1*embLen;
                    indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp1);
                    uint writeVertex;
                    if(len1SmallLen2){
                        writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[1];
                    }else {
                        writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[1];
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[0];
                    }
                    newEmb_g[l] = writeVertex;
                }
                if(len_32<len){
                    uint tmp1 = (len_32+laneId)/embLen;
                    uint tmp2 = (len_32+laneId)-tmp1*embLen;
                    indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp1);
                    uint writeVertex;
                    if(len1SmallLen2){
                        writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[1];
                    }else {
                        writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[1];
                        writeVertex = tmp2<embLen-1?writeVertex:indexs[0];
                    }
                    if(laneId<len-len_32) { newEmb_g[l] = writeVertex; }
                }
                if(laneId==0) { tmp += transsize*embLen; tmpmem_s[64+k] = tmp; }
            }
            uint tmptot=0;
            newInnerLen = 0;
            int newboundryOfNoNeedEval = -1;
            if(notOverMaxRowNum){
                for(int k=boundryOfNoNeedEval+1;k<totcount;++k) {
                    indexs[0] = __shfl_sync(0xffffffff,outerVid,k);

                    if (len1SmallLen2) {
                        predicate = innerVid>0&&innerVid<indexs[0]?1:0;
                        predicate = __ballot_sync(0xffffffff,predicate);
                        if(predicate==0) { newInnerLen = innerLen-((j&0xffffffe0)+transsize); break; }
                        else {
                            predicate = __ffs(predicate)-1;
                            newboundryOfNoNeedEval = k;
                            tmptot += innerLen-(j&0xffffffe0)-predicate;
                            if(laneId==0) { tmpmem_s[32+k] = predicate; }
                        }
                    }else {
                        predicate = innerVid>0&&innerVid>indexs[0]?1:0;
                        predicate = __ballot_sync(0xffffffff,predicate);
                        if(predicate==0) { newInnerLen = innerLen-((j&0xffffffe0)+transsize); neigData1_g += ((j&0xffffffe0)+transsize);  break; }
                        else {
                            predicate = __ffs(predicate)-1;
                            newboundryOfNoNeedEval = k;
                            tmptot += innerLen-(j&0xffffffe0)-predicate;
                            if(laneId==0) { tmpmem_s[32+k] = predicate; }
                        }
                    }
                }
                if(newboundryOfNoNeedEval>=0) {
                    ALLOCSPACE(tmptot,embLen)
                    if(writePos+tmptot>=maxRowNum){
                        notOverMaxRowNum = false;
                        if(laneId==0){
                            tmpmem_s[32] = i>>5;
                            tmpmem_s[33] = j>>5;
                            tmpmem_s[34] = maxRowNum;
                            tmpmem_s[35] = innerLen;
                            tmpmem_s[36] = (boundryOfNoNeedEval+1);
                            tmpmem_s[37] = tmptot;
                        }
                        if(boundryOfNoNeedEval==-1){
                            tmpmem_s[64+laneId] = tmpmem_s[32+laneId];
                            return;
                        }
                    }else{
                        tmptot = newEmb_g-basenewEmb_g;
                        for(int k=boundryOfNoNeedEval+1;k<=newboundryOfNoNeedEval;++k) {
                            indexs[0] = __shfl_sync(0xffffffff,outerVid,k);
                            predicate = tmpmem_s[32+k];
                            uint len = (transsize-predicate)*embLen;
                            uint len_32 = len&0xffffffe0;
                            uint l;
                            for(l=laneId;l<len_32;l=l+32) {
                                uint tmp1 = l/embLen;
                                uint tmp2 = l-tmp1*embLen;
                                indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp1);
                                uint writeVertex;
                                if(len1SmallLen2){
                                    writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
                                    writeVertex = tmp2<embLen-1?writeVertex:indexs[1];
                                }else {
                                    writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[1];
                                    writeVertex = tmp2<embLen-1?writeVertex:indexs[0];
                                }
                                newEmb_g[l] = writeVertex;
                            }
                            if(len_32<len){
                                uint tmp1 = (len_32+laneId)/embLen;
                                uint tmp2 = (len_32+laneId)-tmp1*embLen;
                                indexs[1] = __shfl_sync(0xffffffff,innerVid,tmp1);
                                uint writeVertex;
                                if(len1SmallLen2){
                                    writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[0];
                                    writeVertex = tmp2<embLen-1?writeVertex:indexs[1];
                                }else {
                                    writeVertex = tmp2<embLen-2?tmpmem_s[tmp2]:indexs[1];
                                    writeVertex = tmp2<embLen-1?writeVertex:indexs[0];
                                }
                                if(laneId<len-len_32) { newEmb_g[l] = writeVertex; }
                            }
                            if(laneId==0) { tmpmem_s[64+k] = tmptot+(transsize-predicate)*embLen; }
                            tmptot = tmptot+(transsize-predicate+innerLen-(j&0xffffffe0))*embLen;
                            newEmb_g = basenewEmb_g + tmptot;
                        }
                        boundryOfNoNeedEval = newboundryOfNoNeedEval;
                    }
                }
            }
        }
        if(!notOverMaxRowNum){
            tmpmem_s[64+laneId] = tmpmem_s[32+laneId];
            return;
        }
        innerLen = newInnerLen>0?newInnerLen:innerLen;
    }
    if(laneId==0) { tmpmem_s[66] = 0; }
}

//invoked by extEmb_2V_1src_1and2_sameLabel_NoHash_kernel
//tmpmem_s 96
//this is modified
template <bool isContinue>
__device__ void genExtEmb_2V_1src_1and2_noRestrict_sameLabel_evalEq_withPhase(uint *neigData1_g, uint *neigData2_g,
    uint len1, uint len2, uint *tmpmem_s, uint *equalVertices_s, uint *totWriteRowNum, uint *basenewEmb_g, 
    uint laneId, uint embLen, uint maxRowNum) {

    uint *newEmb_g, i, predicate,tmp;
    uint equalVNum = equalVertices_s[1];
    uint equalVPos = equalVertices_s[2+laneId];
    uint equalVid = laneId<equalVNum?tmpmem_s[equalVPos]:0;
    if(isContinue){ tmp = tmpmem_s[32+laneId]; }
    for(i=0;i<equalVNum;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = equalVPos>equalVid;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate)-(32-equalVNum);
        if(laneId==0){
            tmpmem_s[32+predicate] = equalVPos;
        }
    }
    equalVid = laneId<equalVNum?tmpmem_s[32+laneId]:0;
    if(isContinue){ tmpmem_s[32+laneId] = tmp; }
    if(neigData1_g+len1<=neigData2_g || neigData2_g+len2<=neigData1_g) {
        genExtEmbCore_2V_1and2_noOverlap_withPhase<1,isContinue>(neigData1_g, neigData2_g, len1, len2, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
        return;
    }

    if(!isContinue){
        if(neigData1_g<neigData2_g) {
            uint tmpLen = neigData2_g-neigData1_g;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData1_g, neigData2_g, tmpLen, len2, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            if(tmpmem_s[66] == maxRowNum){
                tmp = tmpmem_s[64];
                tmp = ((tmp<<4) | 1)&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000)|tmp; }
                return;
            }
            neigData1_g = neigData2_g;
            len1 = len1 - tmpLen;
        }else if(neigData2_g<neigData1_g) {
            uint tmpLen = neigData1_g-neigData2_g;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData1_g, neigData2_g, len1, tmpLen, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            if(tmpmem_s[66] == maxRowNum){
                tmp = tmpmem_s[64];
                tmp = ((tmp<<4) | 2)&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000)|tmp; }
                return;
            }
            neigData2_g = neigData1_g;
            len2 = len2 - tmpLen;
        }
        if(len1<len2) {
            uint tmpLen = len2-len1;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData1_g, neigData2_g+len1, len1, tmpLen, tmpmem_s, equalVid, equalVNum, equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            if(tmpmem_s[66] == maxRowNum){
                tmp = tmpmem_s[64];
                tmp = ((tmp<<4) | 3)&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000)|tmp; }
                return;
            }
            len2 = len1;
        }else if(len1>len2){
            uint tmpLen = len1-len2;
            genExtEmbCore_2V_1and2_noOverlap_withPhase<1,false>(neigData1_g+len2, neigData2_g, tmpLen, len2, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            if(tmpmem_s[66] == maxRowNum){
                tmp = tmpmem_s[64];
                tmp = ((tmp<<4) | 4)&0x0000ffff;
                if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000)|tmp; }
                return;
            }
            len1 = len2;
        }
        genExtEmbCore_2V_1and2_noRestrict_fullOverlap_withPhase<false>(neigData1_g,len1,tmpmem_s,equalVid,0,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
        if(tmpmem_s[66] == maxRowNum){
            tmp = tmpmem_s[64];
            tmp = ((tmp<<4) | 5)&0x0000ffff;
            if(laneId==0) { tmpmem_s[64] = (tmpmem_s[64]&0xffff0000)|tmp; }
            return;
        }
    }else{
        uint stopindex = tmpmem_s[64]&0x0000000f;
        if(laneId==0) { tmpmem_s[64] = ((tmpmem_s[64]&0x0000ffff)>>4)| (tmpmem_s[64]&0xffff0000); }
        if(neigData1_g<neigData2_g) {
            uint tmpLen = neigData2_g-neigData1_g;
            if(stopindex==1){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData1_g, neigData2_g, tmpLen, len2, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            }
            neigData1_g = neigData2_g;
            len1 = len1 - tmpLen;
        }else if(neigData2_g<neigData1_g) {
            uint tmpLen = neigData1_g-neigData2_g;
            if(stopindex<=2){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData1_g, neigData2_g, len1, tmpLen, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            }
            neigData2_g = neigData1_g;
            len2 = len2 - tmpLen;
        }
        if(len1<len2) {
            uint tmpLen = len2-len1;
            if(stopindex<=3){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData1_g, neigData2_g+len1, len1, tmpLen, tmpmem_s, equalVid, equalVNum, equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            }
            len2 = len1;
        }else if(len1>len2){
            uint tmpLen = len1-len2;
            if(stopindex<=4){
                genExtEmbCore_2V_1and2_noOverlap_withPhase<1,true>(neigData1_g+len2, neigData2_g, tmpLen, len2, tmpmem_s, equalVid, equalVNum,equalVNum, totWriteRowNum, basenewEmb_g, laneId, embLen,maxRowNum);
            }
            len1 = len2;
        }
        genExtEmbCore_2V_1and2_noRestrict_fullOverlap_withPhase<true>(neigData1_g,len1,tmpmem_s,equalVid,0,equalVNum,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
    }
}

//incoked by extEmb_2V_2src_1L
//needs 96 shared mem
//this is modified
template<bool isContinue>
__device__ void genExtEmb_2V_2src_1and2_noRestrict_sameLabel_evalEq_withPhase(uint *neigData1_g, uint *neigData2_g,
           uint len1, uint len2, uint *tmpmem_s, uint *equalVertices_s, uint *totWriteRowNum,uint *basenewEmb_g,
           uint laneId, uint embLen, uint maxRowNum) {

    uint i,predicate,totcount, *newEmb_g,writePos,tmp,stopindex,loopindex;
    uint equalVNum = equalVertices_s[1];
    uint equalVPos = equalVertices_s[2+laneId];
    uint equalVid = laneId<equalVNum?tmpmem_s[equalVPos]:0;
    if(isContinue) { tmp = tmpmem_s[32+laneId]; }
    for(i=0;i<equalVNum;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = equalVPos>equalVid;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate)-(32-equalVNum);
        if(laneId==0){
            tmpmem_s[32+predicate] = equalVPos;
        }
    }
    equalVid = laneId<equalVNum?tmpmem_s[32+laneId]:0;
    if(isContinue) { tmpmem_s[32+laneId] = tmp; }
    if(isContinue){
        stopindex = tmpmem_s[64+5]&0x0000000f;
        loopindex = (tmpmem_s[64+5]&0xffff0000)>>16;
    }
    if(len1<len2) {
        i=laneId; totcount=0;
        uint outerVid, tot_32 = (len1+31)&0xffffffe0;
        while(totcount==0 && i<tot_32) {
            outerVid = i<len1?neigData1_g[i]:0;
            REARRANGE(outerVid,predicate,totcount,0,equalVNum,64)
            i=i+32;
        }
        if(totcount==0) { if(laneId==0) { tmpmem_s[66]=0;} return; }
        if(!isContinue){
            tmpmem_s[32+laneId] = 0;
            ALLOCSPACE(len2,embLen)
            if(writePos+len2>=maxRowNum){
                if(laneId==0){
                    tmpmem_s[64] = 1;
                    tmpmem_s[65] = len2;
                    tmpmem_s[66] = maxRowNum;
                }
                return;
            }
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,true>(outerVid,neigData2_g,len2,tmpmem_s,equalVid,0,equalVNum,newEmb_g,laneId,embLen);
        }else{
            if(stopindex==1){
                tmpmem_s[32+laneId] = 0;
                ALLOCSPACE(len2,embLen)
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<false,true>(outerVid,neigData2_g,len2,tmpmem_s,equalVid,0,equalVNum,newEmb_g,laneId,embLen);
            }
        }
        uint phaseNum = tmpmem_s[32];
        totcount = totcount - 1;
        if(totcount==0) { if(laneId==0) { tmpmem_s[66]=0;}return; }
        outerVid = __shfl_down_sync(0xffffffff,outerVid,1);
        if(!isContinue){
            ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
            if(writePos+((len2-phaseNum)*totcount)>=maxRowNum){
                if(laneId==0){
                    tmpmem_s[64] = 2;
                    tmpmem_s[65] = ((len2-phaseNum)*totcount);
                    tmpmem_s[66] = maxRowNum;
                }
                return;
            }
        }else{
            if(stopindex<=2){
                ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
            }
        }       
        if(tmpmem_s[32+1+phaseNum-1]!=len2-1) {
            if(laneId==0) { tmpmem_s[32+1+phaseNum] = len2; }
            phaseNum = phaseNum+1;
            //if(laneId==0) { tmpmem_s[32] = phaseNum; }
        }
        if(!isContinue){
            genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,true>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
        }else{
            if(stopindex<=2){
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,true>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
        }
        if(len1>32) {
            if(isContinue){
                if(stopindex==3){
                    i=loopindex+laneId;
                }
            }
            for(;i<tot_32;i=i+32) {
                outerVid = i<len1?neigData1_g[i]:0;
                REARRANGE(outerVid,predicate,totcount,0,equalVNum,64)
                if(totcount==0) { continue; }
                ALLOCSPACE(((len2-phaseNum)*totcount),embLen)
                if(!isContinue){
                    if(writePos+((len2-phaseNum)*totcount)>=maxRowNum){
                        if(laneId==0){
                            tmpmem_s[64] = ((i>>5)<<16)|3;
                            tmpmem_s[65] = ((len2-phaseNum)*totcount);
                            tmpmem_s[66] = maxRowNum;
                        }
                        return;
                    }
                }
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<false,true>(outerVid,neigData2_g,totcount,len2,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
        }
    }else {
        i=laneId; totcount=0;
        uint innerVid, tot_32 = (len2+31)&0xffffffe0;
        while(totcount==0 && i<tot_32) {
            innerVid = i<len2?neigData2_g[i]:0;
            REARRANGE(innerVid,predicate,totcount,0,equalVNum,64)
            i=i+32;
        }
        if(totcount==0) { if(laneId==0) {tmpmem_s[66] = 0;} return; }
        if(!isContinue){
            ALLOCSPACE(len1,embLen)
            tmpmem_s[32+laneId] = 0;
            if(writePos+len1>=maxRowNum){
                if(laneId==0){
                    tmpmem_s[64] = 1;
                    tmpmem_s[65] = len1;
                    tmpmem_s[66] = maxRowNum;
                }
                return;
            }
            genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,true>(innerVid,neigData1_g,len1,tmpmem_s,equalVid,0,equalVNum,newEmb_g,laneId,embLen);
        }else{
            if(stopindex==1){
                ALLOCSPACE(len1,embLen)
                tmpmem_s[32+laneId] = 0;
                genExtEmbCore_2V_1and2_sameLabel_detectPhase<true,true>(innerVid,neigData1_g,len1,tmpmem_s,equalVid,0,equalVNum,newEmb_g,laneId,embLen);
            }
        }
        uint phaseNum = tmpmem_s[32];
        totcount = totcount - 1;
        if(totcount==0){ if(laneId==0) {tmpmem_s[66] = 0;} return; }
        innerVid = __shfl_down_sync(0xffffffff,innerVid,1);
        if(!isContinue){
            ALLOCSPACE(((len1-phaseNum)*totcount),embLen)
            if(writePos+((len1-phaseNum)*totcount)>=maxRowNum){
                if(laneId==0){
                    tmpmem_s[64] = 2;
                    tmpmem_s[65] = ((len1-phaseNum)*totcount);
                    tmpmem_s[66] = maxRowNum;
                }
                return;
            }
        }else{
            if(stopindex<=2){
                ALLOCSPACE(((len1-phaseNum)*totcount),embLen)
            }
        }
        if(tmpmem_s[32+1+phaseNum-1]!=len1-1) {
            if(laneId==0) { tmpmem_s[32+1+phaseNum] = len1; }
            phaseNum = phaseNum+1;
            //if(laneId==0) { tmpmem_s[32] = phaseNum; }
        }
        if(!isContinue){
            genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,true>(innerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
        }else{
            if(stopindex<=2){
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,true>(innerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
        }
        if(len2>32) {
            if(isContinue){
                if(stopindex==3){
                    i=loopindex+laneId;
                }
            }
            for(;i<tot_32;i=i+32) {
                innerVid = i<len2?neigData2_g[i]:0;
                REARRANGE(innerVid,predicate,totcount,0,equalVNum,64)
                if(totcount==0) { continue; }
                ALLOCSPACE(((len1-phaseNum)*totcount),embLen)
                if(!isContinue){
                    if(writePos+((len1-phaseNum)*totcount)>=maxRowNum){
                        if(laneId==0){
                            tmpmem_s[64] = ((i>>5)<<16)|3;
                            tmpmem_s[65] = ((len1-phaseNum)*totcount);
                            tmpmem_s[66] = maxRowNum;
                        }
                        return;
                    }
                }
                genExtEmbCore_2V_1and2_noRestrict_sameLabel_usePhase<true,true>(innerVid,neigData1_g,totcount,len1,tmpmem_s,phaseNum,newEmb_g,laneId,embLen);
            }
        }
    }
    if(laneId==0){
        tmpmem_s[66] = 0;
    }
}

//invoked by extemb_2V_1src_2L, extEmb_2V_2src_2L
//equalVNum1 is the number for extvid1label, equalVNum2 is the number of both extvid1label and extvid2label
//tmpmem_s 96
//this is modified
template<bool isContinue>
__device__ void genExtEmb_2V_1and2_notSameLabel_withPhase(uint *neigData1_g, uint *neigData2_g, uint len1, uint len2, uint *tmpmem_s,
           uint *equalVertices_s, uint *totWriteRowNum, uint *basenewEmb_g,uint laneId, uint embLen,uint maxRowNum) {
    
    uint i, predicate,tmp;
    uint equalVNum1 = equalVertices_s[0];
    uint equalVNum2 = equalVertices_s[1];
    uint equalVPos = equalVertices_s[2+laneId];
    uint equalVid = laneId<equalVNum2?tmpmem_s[equalVPos]:0;
    if(isContinue){
        tmp = tmpmem_s[32+laneId];
    }
    for(i=0;i<equalVNum1;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = laneId<equalVNum1?equalVPos>equalVid:0;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate);
        if(laneId==0){
            tmpmem_s[32+predicate] = equalVPos;
        }
    }
    for(i=equalVNum1;i<equalVNum2;++i){
        equalVPos = __shfl_sync(0xffffffff,equalVid,i);
        predicate = laneId>=equalVNum1 && laneId<equalVNum2?equalVPos>equalVid:0;
        predicate = __ballot_sync(0xffffffff, predicate);
        predicate = __popc(predicate);
        if(laneId==0){
            tmpmem_s[32+equalVNum1+predicate] = equalVPos;
        }
    }
    equalVid = laneId<equalVNum2?tmpmem_s[32+laneId]:0;
    if(isContinue){
        tmpmem_s[32+laneId] = tmp;
    }
    genExtEmbCore_2V_1and2_noOverlap_withPhase<2,isContinue>(neigData1_g,neigData2_g,len1,len2,tmpmem_s,equalVid,equalVNum1,equalVNum2,totWriteRowNum,basenewEmb_g,laneId,embLen,maxRowNum);
}




//return the number of vertices that lower than or equal to greatV
__device__ uint findLessVPosNoIndex(uint *neigData_g, uint neigLen, uint lessV, uint laneId) {
    uint tmp2, i, predicate = 0;
    tmp2 = ((neigLen + 31) & 0xffffffe0);
    for (i = laneId; i < tmp2; i = i + 32) {
        predicate = i < neigLen ? (neigData_g[i] < lessV) : 0;
        predicate = __ballot_sync(0xffffffff, predicate);
        if (predicate < 0xffffffff) { predicate = __popc(predicate); return (i & 0xffffffe0) + predicate; }
    }
    return neigLen;
}

__device__ uint findLessVPosWithIndex(uint *neigData_g, uint neigLen, uint lessV, uint laneId, uint indexNum){
    uint tmp2, i, predicate=0, upperLimit,lowerLimit;
    upperLimit = laneId<indexNum?neigData_g[laneId]:0;
    lowerLimit = __shfl_up_sync(0xffffffff,upperLimit,1);
    if(laneId==0){ lowerLimit = neigData_g[indexNum]-1; } //neigData_g[indexNum] is the first and smallest vertex of neighbors
    predicate = (lowerLimit<lessV && lessV<=upperLimit);
    predicate = __ballot_sync(0xffffffff,predicate);
    if(predicate==0){ return 0; }
    tmp2 = __ffs(predicate);
    uint blockSize = neigLen<=32*VBLOCKSIZE?VBLOCKSIZE:(neigLen>>5);
    lowerLimit = tmp2*blockSize;
    blockSize = tmp2==(indexNum-1)?neigLen-blockSize*tmp2:blockSize;
    upperLimit = (blockSize+31)&0xffffffe0;
    for(i=laneId;i<upperLimit;i=i+32){
        predicate = i<blockSize?(neigData_g[indexNum+lowerLimit+i]<lessV):0;
        predicate = __ballot_sync(0xffffffff,predicate);
        if(predicate<0xffffffff){ predicate = __popc(predicate); return lowerLimit+(i&0xffffffe0)+predicate; }
    }
    return lowerLimit+blockSize;
}

#define INIT_VARS_SHAREDMEM_NOAUX(SIZE)                                         \
    laneId = threadIdx.x & 31;                                                  \
    gridWarpNum = (gridDim.x * blockDim.x)>>5;                                       \
    warpIdInBlock = threadIdx.x >> 5;                                           \
    __shared__ uint neigV[WARPPERBLOCK][SIZE];                                  \
    __shared__ uint indexForIndex[256*2+1+1+64+1];                                    \
    for(i=threadIdx.x;i<intervalNum*2+1+1;i=i+blockDim.x){                        \
        indexForIndex[i] = edgeLabelPartition[i];                               \
    }                                                                           \
    i = blockIdx.x * blockDim.x + threadIdx.x;                                  \
    i >>= 5;                                                                    \
    edgeLabelPartition = edgeLabelPartition+intervalNum*2+1+1;\
    if(threadIdx.x==0){ indexForIndex[256*2+66] = 0; }

//before this macro, edgeLabelPartition points to array0, after this macro
//edgeLabelPartition points to array1
#define INIT_VARS_SHAREDMEM(SIZE)                                               \
    laneId = threadIdx.x & 31;                                                  \
    gridWarpNum = (gridDim.x * blockDim.x)>>5;                                  \
    warpIdInBlock = threadIdx.x >> 5;                                           \
    __shared__ uint neigV[WARPPERBLOCK][SIZE];                                  \
    __shared__ uint indexForIndex[256*2+1+1+64+1];                                \
    for(i=threadIdx.x;i<intervalNum*2+1+1;i=i+blockDim.x){                      \
        indexForIndex[i] = edgeLabelPartition[i];                               \
    }                                                                           \
    for(i=threadIdx.x;i<64;i=i+blockDim.x){                                     \
        indexForIndex[256*2+1+1+i] = auxArray[i];                               \
    }                                                                           \
    i = blockIdx.x * blockDim.x + threadIdx.x;                                  \
    i >>= 5;                                                                    \
    edgeLabelPartition = edgeLabelPartition+intervalNum*2+1+1;\
    if(threadIdx.x==0){ indexForIndex[256*2+66] = 0; }

//init: generate embeddings of the first partial pattern
//ext: extension
//genRec: generate positions of edge label--end vertex label combination
//edgeLabelPart: the position of array2
//vLabel: label id of each vertex
//totWriteRowNum is a global number, all lane 0 accumulate their number of generated embeddings onto this variable
//recordPos: store adresses of neighbor into recordPos. recordPos[i*2]=0,recordPos[i*2+1]=0 means the label doesn't fit, we do not
//have the change to find its neighbor address for this label. recordPos[i*2]=1,recordPos[i*2+1]=0 means this svid does not have
//neighbors of this label
//restrictFlag:0-7 bits, extvid1<v[pos]; 8-15 bits, extvid2<v[pos]; 16-23 bits, is extvid1<extvid2, 0, 1; 0xff means no restrict
//indexForIndex[0:2*256] are interval indexs, indexForIndex[2*256] and indexForIndex[2*256+1] are labels for extvid1 and extvid2
//indexForIndex[2*256+2] and indexForIndex[2*256+3] are positions of lessvid for extvid1 and extvid2
//the format for indexForIndex[0:2*256] is vs1,vs2,vs3,len1,len2,len3. len1 is the numberr of vertices in interval 1, len2
//is the number of vertices in all previsous intervals (including interval 2).
//neighborsData is the address of array4
//this is modified
template<bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict, bool isExt1and2SameLabel, bool isRecord1, bool isRecord2>
__global__ void initEmb_3V_1and2_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, 
    uint *neighborsData, uint *recordPos1, uint *recordPos2, uint *newEmb, uint intervalNum, uint partialRowNum, 
    uint svidlabel,uint evid1label,uint evid2label, bool isContinue, uint maxRowNum){

    uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM_NOAUX(32)
    __syncthreads();
    uint intervalStart = laneId,predicate;
    if(isContinue){
        uint writeEle = laneId<7?newEmb[(maxRowNum+i)*7+laneId]:0;
        neigV[warpIdInBlock][laneId] = writeEle; 
        uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][0];
        uint neigLen1 = neigV[warpIdInBlock][1];
        uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][2];
        uint neigLen2 = neigV[warpIdInBlock][3];
        i = neigV[warpIdInBlock][4];
        intervalStart = neigV[warpIdInBlock][5]+laneId;
        uint svid = neigV[warpIdInBlock][6];
        if(laneId==0) { neigV[warpIdInBlock][0] = svid;}
        if(isExt1and2SameLabel){
            if (isExt1and2Restrict) {
                genInitEmb_3V_1and2_restric(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,svid,maxRowNum);
            } else {
                genInitEmb_3V_1and2_noRestrict_sameLabel(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
            }
        }else{
            genInitEmb_3V_1and2_notsamelabel(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
        }
        i=i+gridWarpNum;
    }
    while (i < partialRowNum){
        if(isRecord1){
            if(laneId<2) { recordPos1[i*2+laneId] = 0; }
        }
        if(isRecord2){
            if(laneId<2) { recordPos2[i*2+laneId] = 0; }
        }
        uint lowerLimit, upperLimit, svid;
        uint len_32 = (intervalNum+31)&0xffffffe0;
        for(j=intervalStart;j<len_32;j=j+32){
            upperLimit = indexForIndex[j+intervalNum+1];
            lowerLimit = indexForIndex[j+intervalNum];
            //i+1 because vid starts from 1
            predicate = j<intervalNum?(i+1>lowerLimit && i+1<=upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0) {
                upperLimit=i-lowerLimit;
                lowerLimit=indexForIndex[j];
                svid=lowerLimit+upperLimit;
                intervalStart = j;
                break;
            }
        }
        if(predicate>0){
            uint tmpIndex = __ffs(predicate);
            svid = __shfl_sync(0xffffffff,svid,tmpIndex-1);
        }
        else{ /*std::cout<<"wrong"<<std::endl;*/ return; }

        if(vLabel[svid]!=svidlabel){ 
            i=i+gridWarpNum; 
            continue; 
        }

        if(laneId==0) { neigV[warpIdInBlock][0] = svid;}
        uint *neigAddr = neighborsData + edgeLabelPartition[i];
        uint neigLen = edgeLabelPartition[i+1]-edgeLabelPartition[i];
        lowerLimit = 0; 
        upperLimit = neigLen;
        uint lessThan, greatThan;
        if(isExt1and2SameLabel) {
            uint found;
            FIND_LABEL_LIMIT(neigAddr,lowerLimit,upperLimit,evid1label)
            neigLen = upperLimit - lowerLimit;
            if (neigLen == 0) { 
                if(isRecord1){ 
                    if (laneId==0) {
                        recordPos1[i*2]=1;recordPos1[i*2+1]=0;
                    } 
                } 
                i=i+gridWarpNum; 
                continue; 
            }
            uint indexNum = 0,neigNum;
            if(neigLen>VBLOCKSIZE+2 && neigLen<=32*VBLOCKSIZE+32) {indexNum=(neigLen+VBLOCKSIZE)/(VBLOCKSIZE+1);}
            else if (neigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }
            neigAddr = neigAddr + lowerLimit;//address starts from index terms
            neigLen = neigLen - indexNum;
            //recorded positon starts from index terms, recorded length is the length of neghbors (not inlcude index terms)
            if (isRecord1) { if(laneId==0) {recordPos1[i*2]=lowerLimit;recordPos1[i*2+1]=neigLen;} }
            if (isExt1Restrict && isExt2Restrict) {
                neigNum = neigLen;
                if (indexNum == 0) { neigLen = findLessVPosNoIndex(neigAddr, neigNum, svid, laneId); }
                else { neigLen = findLessVPosWithIndex(neigAddr, neigNum, svid, laneId, indexNum); }
                if (neigLen == 0) { i=i+gridWarpNum;continue; }
                neigNum = neigLen;
                if (isExt1and2Restrict) {
                    genInitEmb_3V_1and2_restric(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,svid,maxRowNum);
                } else {
                    genInitEmb_3V_1and2_noRestrict_sameLabel(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
                }
            } else if (isExt1Restrict) {
                neigNum = neigLen;
                if (indexNum == 0) { neigLen = findLessVPosNoIndex(neigAddr, neigNum, svid, laneId); }
                else { neigLen = findLessVPosWithIndex(neigAddr, neigNum, svid, laneId, indexNum); }
                if (neigLen == 0) { i=i+gridWarpNum;continue; }
                if (isExt1and2Restrict) {
                    genInitEmb_3V_1and2_restric(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,svid,maxRowNum);
                } else {
                    genInitEmb_3V_1and2_noRestrict_sameLabel(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
                }
            } else if (isExt2Restrict) {
                if (indexNum == 0) { neigNum = findLessVPosNoIndex(neigAddr, neigLen, svid, laneId); }
                else { neigNum = findLessVPosWithIndex(neigAddr, neigLen, svid, laneId, indexNum); }
                if (neigNum == 0) { i=i+gridWarpNum;continue; }
                if (isExt1and2Restrict) {
                    genInitEmb_3V_1and2_restric(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,svid,maxRowNum);
                } else {
                    genInitEmb_3V_1and2_noRestrict_sameLabel(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
                }
            } else {
                neigNum = neigLen;
                if (isExt1and2Restrict) {
                    genInitEmb_3V_1and2_restric(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,svid,maxRowNum);
                } else {
                    genInitEmb_3V_1and2_noRestrict_sameLabel(neigAddr+indexNum,neigAddr+indexNum,neigLen,neigNum,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
                }
            }
            if(neigV[warpIdInBlock][0]==maxRowNum){
                uint index = blockIdx.x*blockDim.x+threadIdx.x;
                uint writeEle = neigAddr+indexNum-neighborsData;
                if(laneId==1){
                    writeEle = neigLen;
                }else if(laneId==2){
                    writeEle = neigAddr+indexNum-neighborsData;
                }else if(laneId==3){
                    writeEle = neigNum;
                }else if(laneId==4){
                    writeEle = i;
                }else if(laneId==5){
                    writeEle = intervalStart>>5;
                }else if(laneId==6){
                    writeEle = svid;
                }
                if(laneId<7){
                    newEmb[(maxRowNum+index)*7+laneId] = writeEle;
                }
                writeEle = neigV[warpIdInBlock][1];
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,writeEle);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }else {
            uint upperLimit2 = upperLimit, lowerLimit2 = lowerLimit, found;
            FIND_LABEL_LIMIT(neigAddr,lowerLimit,upperLimit,evid1label)
            neigLen = upperLimit - lowerLimit;
            if (neigLen == 0) { if(isRecord1){ if (laneId==0) {recordPos1[i*2]=1;recordPos1[i*2+1]=0;} } i=i+gridWarpNum; continue; }
            FIND_LABEL_LIMIT(neigAddr, lowerLimit2, upperLimit2, evid2label)
            uint neigLen2 = upperLimit2 - lowerLimit2;
            if (neigLen2 == 0) { if(isRecord2){ if (laneId==0) {recordPos2[i*2]=1;recordPos2[i*2+1]=0;} } i=i+gridWarpNum; continue; }
            uint indexNum = 0,neigNum, indexNum2=0,neigNum2;
            if(neigLen>VBLOCKSIZE+2 && neigLen<=32*VBLOCKSIZE+32) {indexNum=(neigLen+VBLOCKSIZE)/(VBLOCKSIZE+1);}
            else if (neigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }
            if(neigLen2>VBLOCKSIZE+2 && neigLen2<=32*VBLOCKSIZE+32) {indexNum2=(neigLen2+VBLOCKSIZE)/(VBLOCKSIZE+1);}
            else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }

            uint *neigAddr2 = neigAddr + lowerLimit2;
            neigLen2 = neigLen2 - indexNum2;
            neigAddr = neigAddr + lowerLimit;
            neigLen = neigLen - indexNum;
            if (isRecord1) { if(laneId==0) {recordPos1[i*2]=lowerLimit;recordPos1[i*2+1]=neigLen;} }
            if (isRecord2) { if(laneId==0) {recordPos2[i*2]=lowerLimit2;recordPos2[i*2+1]=neigLen2;} }
            if (isExt1Restrict) {
                if (indexNum == 0) { neigLen = findLessVPosNoIndex(neigAddr, neigLen, svid, laneId); }
                else { neigLen = findLessVPosWithIndex(neigAddr, neigLen, svid, laneId, indexNum); }
                if (neigLen == 0) { i=i+gridWarpNum;continue; }
                genInitEmb_3V_1and2_notsamelabel(neigAddr+indexNum,neigAddr2+indexNum2,neigLen,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
            } else if (isExt2Restrict) {
                if (indexNum2 == 0) { neigLen2 = findLessVPosNoIndex(neigAddr2, neigLen2, svid, laneId); }
                else { neigLen2 = findLessVPosWithIndex(neigAddr2, neigLen2, svid, laneId, indexNum2); }
                if (neigLen2 == 0) { i=i+gridWarpNum;continue; }
                genInitEmb_3V_1and2_notsamelabel(neigAddr+indexNum,neigAddr2+indexNum2,neigLen,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
            } else {
                genInitEmb_3V_1and2_notsamelabel(neigAddr+indexNum,neigAddr2+indexNum2,neigLen,neigLen2,neigV[warpIdInBlock],newEmb,totWriteRowNum,laneId,maxRowNum);
            }
            if(neigV[warpIdInBlock][0]==maxRowNum){
                uint index = blockIdx.x*blockDim.x+threadIdx.x;
                uint writeEle = neigAddr+indexNum-neighborsData;
                if(laneId==1){
                    writeEle = neigLen;
                }else if(laneId==2){
                    writeEle = neigAddr2+indexNum2-neighborsData;
                }else if(laneId==3){
                    writeEle = neigLen2;
                }else if(laneId==4){
                    writeEle = i;
                }else if(laneId==5){
                    writeEle = intervalStart>>5;
                }else if(laneId==6){
                    writeEle = svid;
                }
                if(laneId<7){
                    newEmb[(maxRowNum+index)*7+laneId] = writeEle;
                }
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,neigLen*neigLen2);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }
        i = i+gridWarpNum;
    }
}

//this is modified
template<bool isExtRestrict, bool isRecord>
__global__ void initEmb_2V_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, uint *neighborsData,uint *recordPos,
    uint *newEmb, uint intervalNum, uint partialRowNum, uint svidlabel,uint evidlabel, bool isContinue,uint maxRowNum){

    uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM_NOAUX(32)
    __syncthreads();
    uint intervalStart = laneId;
    if(isContinue){
        uint writeEle = laneId<5?newEmb[(maxRowNum+i)*5+laneId]:0;
        neigV[warpIdInBlock][laneId] = writeEle; 
        i = neigV[warpIdInBlock][2];
        uint svid = neigV[warpIdInBlock][3];
        intervalStart = neigV[warpIdInBlock][4]+laneId;
        uint *neigAddr = neighborsData+neigV[warpIdInBlock][0];
        uint neigLen = neigV[warpIdInBlock][1];
        genInitEmb_2V(neigAddr,neigLen,newEmb,totWriteRowNum,laneId,svid,neigV[warpIdInBlock],maxRowNum);
        i=i+gridWarpNum;
    }
    while (i < partialRowNum){
        uint lowerLimit, upperLimit, svid,predicate;
        uint len_32 = (intervalNum+31)&0xffffffe0;
        for(j=intervalStart;j<len_32;j=j+32){
            upperLimit = indexForIndex[j+intervalNum+1];
            lowerLimit = indexForIndex[j+intervalNum];
            //i+1 because vid starts from 1
            predicate = j<intervalNum?(i+1>lowerLimit && i+1<=upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0) {
                upperLimit=i-lowerLimit;
                lowerLimit=indexForIndex[j];
                svid=lowerLimit+upperLimit;
                intervalStart = j;
                break;
            }
        }
        if(predicate>0){
            uint tmpIndex = __ffs(predicate);
            svid = __shfl_sync(0xffffffff,svid,tmpIndex-1);
        }else{ return; }
        if(vLabel[svid]!=svidlabel){ i=i+gridWarpNum; continue; }
        if(laneId==0) { neigV[warpIdInBlock][0] = svid;}
        uint *neigAddr = neighborsData + edgeLabelPartition[i];
        uint neigLen = edgeLabelPartition[i+1]-edgeLabelPartition[i],found;

        lowerLimit = 0;
        upperLimit = neigLen;
        uint lessThan,greatThan;
        FIND_LABEL_LIMIT(neigAddr,lowerLimit, upperLimit, evidlabel)
        neigLen = upperLimit - lowerLimit;
        if (neigLen == 0) { if(isRecord){ if (laneId==0) {recordPos[i*2]=1;recordPos[i*2+1]=0;} } i=i+gridWarpNum; continue; }
        uint indexNum = 0,neigNum;
        if(neigLen>VBLOCKSIZE+2 && neigLen<=32*VBLOCKSIZE+32) {indexNum=(neigLen+VBLOCKSIZE)/(VBLOCKSIZE+1);}
        else if (neigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }
        neigAddr = neigAddr + lowerLimit;
        neigLen = neigLen - indexNum;
        if (isRecord) { if(laneId==0) {recordPos[i*2]=lowerLimit;recordPos[i*2+1]=neigLen;} }
        if(isExtRestrict) {
            if (indexNum == 0) { neigNum = findLessVPosNoIndex(neigAddr, neigLen, svid, laneId); }
            else { neigNum = findLessVPosWithIndex(neigAddr, neigLen, svid, laneId, indexNum); }
            if (neigNum == 0) { i=i+gridWarpNum; continue; }
            genInitEmb_2V(neigAddr+indexNum,neigNum,newEmb,totWriteRowNum,laneId,svid,neigV[warpIdInBlock],maxRowNum);
            if(neigV[warpIdInBlock][0]==maxRowNum){
                uint index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
                uint writeEle = neigAddr+indexNum-neighborsData;
                if(laneId==1){
                    writeEle = neigNum;
                }else if(laneId==2){
                    writeEle = i;
                }else if(laneId==3){
                    writeEle = svid;
                }else if(laneId==4){
                    writeEle = intervalStart>>5;
                }
                if(laneId<5){
                    newEmb[(maxRowNum+index)*5+laneId]=writeEle;
                }
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,neigNum);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }else{
            genInitEmb_2V(neigAddr+indexNum,neigLen,newEmb,totWriteRowNum,laneId,svid,neigV[warpIdInBlock],maxRowNum);
            if(neigV[warpIdInBlock][0]==maxRowNum){
                uint index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
                uint writeEle = neigAddr+indexNum-neighborsData;
                if(laneId==1){
                    writeEle = neigLen;
                }else if(laneId==2){
                    writeEle = i;
                }else if(laneId==3){
                    writeEle = svid;
                }else if(laneId==4){
                    writeEle = intervalStart>>5;
                }
                if(laneId<5){
                    newEmb[(maxRowNum+index)*5+laneId]= writeEle;
                }
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,neigLen);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }
        i = i+gridWarpNum;
    }
}


//this is modified
template<bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict,bool isRecord, bool useRecord>
__global__ void extEmb_2V_2src_1and2_sameLabel_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, 
    uint *neighborsData, uint *recordPos, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, 
    uint partialRowNum, uint embLen, bool isContinue, uint maxRowNum) {

    uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM(96)
    __syncthreads();
    if(isContinue){
        if(isExt1and2Restrict){
            uint writeEle = laneId<9?newEmb[(maxRowNum+i)*9+laneId]:0;
            neigV[warpIdInBlock][32+laneId] = writeEle; 
            i = neigV[warpIdInBlock][32+4];
            j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
            neigV[warpIdInBlock][laneId] = j;
            uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][32+0];
            uint neigLen1 = neighborsData[32+1];
            uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][32+2];
            uint neigLen2 = neighborsData[32+3];
            if(neigLen1<neigLen2){
                genExtEmb_2V_2src_1and2_restrict<true,true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,newEmb,laneId,embLen,maxRowNum);
            }else{
                genExtEmb_2V_2src_1and2_restrict<false,true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,newEmb,laneId,embLen,maxRowNum);
            }
        }else{
            uint writeEle = newEmb[(maxRowNum+i)*38+laneId];
            neigV[warpIdInBlock][32+laneId] = writeEle;
            writeEle = laneId<6?newEmb[(maxRowNum+i)*38+32+laneId]:0;
            neigV[warpIdInBlock][64+laneId] = writeEle;
            i = neigV[warpIdInBlock][64+4];
            j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
            neigV[warpIdInBlock][laneId] = j;
            uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][64+0];
            uint neigLen1 = neighborsData[64+1];
            uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][64+2];
            uint neigLen2 = neighborsData[64+3];
            genExtEmb_2V_2src_1and2_noRestrict_sameLabel_evalEq_withPhase<true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        }
        i = i + gridWarpNum;
    }
    while (i < partialRowNum) {
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        j = __ballot_sync(0xffffffff,j);
        if(j<0xffffffff) { i=i+gridWarpNum; continue; }

        uint svid1 = indexForIndex[256*2+1+1+4];
        uint svid2 = indexForIndex[256*2+1+1+5];
        svid1 = neigV[warpIdInBlock][svid1];
        svid2 = neigV[warpIdInBlock][svid2];
        uint lowerLimit = 0, upperLimit, predicate = 0, index;
        uint len = (intervalNum+31)&0xffffffe0;
        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid1>=lowerLimit && svid1<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid1-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        uint baseNum = edgeLabelPartition[index];
        uint *neigAddr1 = neighborsData+baseNum;
        uint neigLen1 = edgeLabelPartition[index+1]-baseNum;
        uint indexNum1 = 0;
        if(useRecord){
            uint distance = recordPos[index*2];
            uint tmpNeigLen1 = recordPos[index*2+1];
            if(distance==0 && tmpNeigLen1==0){
                uint evidlabel = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen1;
                FIND_LABEL_LIMIT(neigAddr1, lowerLimit, upperLimit, evidlabel)
                neigLen1 = upperLimit - lowerLimit;
                if (neigLen1 == 0) {
                    if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
                neigAddr1 = neigAddr1 + lowerLimit;
                neigLen1 = neigLen1 - indexNum1;
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen1; } }
            }else if(distance==1 && tmpNeigLen1==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr1 = neigAddr1+distance;
                neigLen1 = tmpNeigLen1;
            }
        }else {
            uint evidlabel = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen1;
            FIND_LABEL_LIMIT(neigAddr1, lowerLimit, upperLimit, evidlabel)
            neigLen1 = upperLimit - lowerLimit;
            if (neigLen1 == 0) {
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
            neigAddr1 = neigAddr1 + lowerLimit;
            neigLen1 = neigLen1 - indexNum1;
            if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen1; } }
        }
        if(isExt1Restrict){
            uint lessvid = indexForIndex[2*256+1+1+2];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum1 == 0) { neigLen1 = findLessVPosNoIndex(neigAddr1, neigLen1, lessvid, laneId); }
            else { neigLen1 = findLessVPosWithIndex(neigAddr1, neigLen1, lessvid, laneId, indexNum1); }
            if (neigLen1 == 0) { i=i+gridWarpNum; continue; }
        }



        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid2>=lowerLimit && svid2<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid2-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        baseNum = edgeLabelPartition[index];
        uint *neigAddr2 = neighborsData+baseNum;
        uint neigLen2 = edgeLabelPartition[index+1]-baseNum;
        uint indexNum2 = 0;
        if(useRecord){
            uint distance = recordPos[index*2];
            uint tmpNeigLen2 = recordPos[index*2+1];
            if(distance==0 && tmpNeigLen2==0){
                uint evidlabel = indexForIndex[2*256+1+1+1],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen2;
                FIND_LABEL_LIMIT(neigAddr2, lowerLimit, upperLimit, evidlabel)
                neigLen2 = upperLimit - lowerLimit;
                if (neigLen2 == 0) {
                    if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
                neigAddr2 = neigAddr2 + lowerLimit;
                neigLen2 = neigLen2 - indexNum2;
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen2; } }
            }else if(distance==1 && tmpNeigLen2==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr2 = neigAddr2+distance;
                neigLen2 = tmpNeigLen2;
            }
        }else {
            uint evidlabel = indexForIndex[2*256+1+1+1],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen2;
            FIND_LABEL_LIMIT(neigAddr2, lowerLimit, upperLimit, evidlabel)
            neigLen2 = upperLimit - lowerLimit;
            if (neigLen2 == 0) {
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
            neigAddr2 = neigAddr2 + lowerLimit;
            neigLen2 = neigLen2 - indexNum2;
            if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen2; } }
        }
        if(isExt2Restrict){
            uint lessvid = indexForIndex[2*256+1+1+3];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum2 == 0) { neigLen2 = findLessVPosNoIndex(neigAddr2, neigLen2, lessvid, laneId); }
            else { neigLen2 = findLessVPosWithIndex(neigAddr2, neigLen2, lessvid, laneId, indexNum2); }
            if (neigLen2 == 0) { i=i+gridWarpNum; continue; }
        }
        if(isExt1and2Restrict){
            if(neigLen1<neigLen2){
                genExtEmb_2V_2src_1and2_restrict<true,false>(neigAddr1+indexNum1,neigAddr2+indexNum2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,newEmb,laneId,embLen,maxRowNum);
            }else{
                genExtEmb_2V_2src_1and2_restrict<false,false>(neigAddr1+indexNum1,neigAddr2+indexNum2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,newEmb,laneId,embLen,maxRowNum);
            }
            if(neigV[warpIdInBlock][66]==maxRowNum){
                index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
                uint writeEle = neigAddr1+indexNum1-neighborsData;
                if(laneId==1){ writeEle = neigLen1; }
                else if(laneId==2){ writeEle = neigAddr2+indexNum2-neighborsData; }
                else if(laneId==3){ writeEle = neigLen2; }
                else if(laneId==4){ writeEle = i; }
                else if(laneId==5){ writeEle = neigV[warpIdInBlock][32]; }
                else if(laneId==6){ writeEle = neigV[warpIdInBlock][33]; }
                else if(laneId==7){ writeEle = neigV[warpIdInBlock][35]; }
                else if(laneId==8){ writeEle = neigV[warpIdInBlock][36]; }
                if(laneId<9){
                    newEmb[(maxRowNum+index)*9+laneId] = writeEle;
                }
                writeEle = neigV[warpIdInBlock][37];
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,writeEle);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }else{
            genExtEmb_2V_2src_1and2_noRestrict_sameLabel_evalEq_withPhase<false>(neigAddr1+indexNum1,neigAddr2+indexNum2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
            if(neigV[warpIdInBlock][66]==maxRowNum){
                index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
                uint writeEle = neigAddr1+indexNum1-neighborsData;
                if(laneId==1){ writeEle = neigLen1; }
                else if(laneId==2){ writeEle = neigAddr2+indexNum2-neighborsData; }
                else if(laneId==3){ writeEle = neigLen2; }
                else if(laneId==4){ writeEle = i; }
                else if(laneId==5){ writeEle = neigV[warpIdInBlock][64];}
                newEmb[(maxRowNum+index)*38+laneId] = neigV[warpIdInBlock][32+laneId];
                if(laneId<6){
                    newEmb[(maxRowNum+index)*38+32+laneId] = writeEle;
                }
                writeEle = neigV[warpIdInBlock][65];
                if(laneId==0){
                    atomicAdd(indexForIndex+256*2+66,writeEle);
                }
                __syncthreads();
                if(threadIdx.x==0){
                    atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
                }
                break;
            }
        }

        i = i + gridWarpNum;
    }
}

//this is modified
template<bool isExt1Restrict,bool isExt2Restrict,bool isRecord1,bool isRecord2,bool useRecord1,bool useRecord2>
__global__ void extEmb_2V_2src_1and2_notSameLabel_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, 
    uint *neighborsData, uint *recordPos1, uint *recordPos2, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, 
    uint partialRowNum, uint embLen,bool isContinue, uint maxRowNum) {

    uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM(96)
    __syncthreads();
    if(isContinue){
        uint writeEle = laneId<6?newEmb[(maxRowNum+i)*38+laneId]:0;
        neigV[warpIdInBlock][laneId+64] = writeEle; 
        neigV[warpIdInBlock][32+laneId] = newEmb[(maxRowNum+i)*38+6+laneId];
        i = neigV[warpIdInBlock][64+4];
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][64+0];
        uint neigLen1 = neighborsData[64+1];
        uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][64+2];
        uint neigLen2 = neighborsData[64+3];
        genExtEmb_2V_1and2_notSameLabel_withPhase<true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        i = i + gridWarpNum;
    }
    while (i < partialRowNum) {
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        j = __ballot_sync(0xffffffff,j);
        if(j<0xffffffff) { i=i+gridWarpNum; continue; }

        uint svid1 = indexForIndex[256*2+1+1+4];
        uint svid2 = indexForIndex[256*2+1+1+5];
        svid1 = neigV[warpIdInBlock][svid1];
        svid2 = neigV[warpIdInBlock][svid2];
        uint lowerLimit = 0, upperLimit, predicate = 0, index;
        uint len = (intervalNum+31)&0xffffffe0;
        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid1>=lowerLimit && svid1<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid1-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        uint baseNum = edgeLabelPartition[index];
        uint *neigAddr1 = neighborsData+baseNum;
        uint neigLen1 = edgeLabelPartition[index+1]-baseNum;
        uint indexNum1 = 0;
        if(useRecord1){
            uint distance = recordPos1[index*2];
            uint tmpNeigLen1 = recordPos1[index*2+1];
            if(distance==0 && tmpNeigLen1==0){
                uint evid1label = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen1;
                FIND_LABEL_LIMIT(neigAddr1, lowerLimit, upperLimit, evid1label)
                neigLen1 = upperLimit - lowerLimit;
                if (neigLen1 == 0) {
                    if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = 1; recordPos1[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
                neigAddr1 = neigAddr1 + lowerLimit;
                neigLen1 = neigLen1 - indexNum1;
                if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = lowerLimit; recordPos1[index * 2 + 1] = neigLen1; } }
            }else if(distance==1 && tmpNeigLen1==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr1 = neigAddr1+distance;
                neigLen1 = tmpNeigLen1;
            }
        }else {
            uint evid1label = indexForIndex[2 * 256+1+1+0],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen1;
            FIND_LABEL_LIMIT(neigAddr1, lowerLimit, upperLimit, evid1label)
            neigLen1 = upperLimit - lowerLimit;
            if (neigLen1 == 0) {
                if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = 1; recordPos1[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
            neigAddr1 = neigAddr1 + lowerLimit;
            neigLen1 = neigLen1 - indexNum1;
            if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = lowerLimit; recordPos1[index * 2 + 1] = neigLen1; } }
        }
        if(isExt1Restrict){
            uint lessvid = indexForIndex[2*256+1+1+2];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum1 == 0) { neigLen1 = findLessVPosNoIndex(neigAddr1, neigLen1, lessvid, laneId); }
            else { neigLen1 = findLessVPosWithIndex(neigAddr1, neigLen1, lessvid, laneId, indexNum1); }
            if (neigLen1 == 0) { i=i+gridWarpNum; continue; }
        }

        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid2>=lowerLimit && svid2<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid2-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        baseNum = edgeLabelPartition[index];
        uint *neigAddr2 = neighborsData+baseNum;
        uint neigLen2 = edgeLabelPartition[index+1]-baseNum;
        uint indexNum2 = 0;
        if(useRecord2){
            uint distance = recordPos2[index*2];
            uint tmpNeigLen2 = recordPos2[index*2+1];
            if(distance==0 && tmpNeigLen2==0){
                uint evid2label = indexForIndex[2 * 256+1+1+1],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen2;
                FIND_LABEL_LIMIT(neigAddr2, lowerLimit, upperLimit, evid2label)
                neigLen2 = upperLimit - lowerLimit;
                if (neigLen2 == 0) {
                    if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = 1; recordPos2[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
                neigAddr2 = neigAddr2 + lowerLimit;
                neigLen2 = neigLen2 - indexNum2;
                if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = lowerLimit; recordPos2[index * 2 + 1] = neigLen2; } }
            }else if(distance==1 && tmpNeigLen2==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr2 = neigAddr2+distance;
                neigLen2 = tmpNeigLen2;
            }
        }else {
            uint evid2label = indexForIndex[2*256+1+1+1],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen2;
            FIND_LABEL_LIMIT(neigAddr2, lowerLimit, upperLimit, evid2label)
            neigLen2 = upperLimit - lowerLimit;
            if (neigLen2 == 0) {
                if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = 1; recordPos2[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
            neigAddr2 = neigAddr2 + lowerLimit;
            neigLen2 = neigLen2 - indexNum2;
            if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = lowerLimit; recordPos2[index * 2 + 1] = neigLen2; } }
        }
        if(isExt2Restrict){
            uint lessvid = indexForIndex[2*256+1+1+3];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum2 == 0) { neigLen2 = findLessVPosNoIndex(neigAddr2, neigLen2, lessvid, laneId); }
            else { neigLen2 = findLessVPosWithIndex(neigAddr2, neigLen2, lessvid, laneId, indexNum2); }
            if (neigLen2 == 0) { i=i+gridWarpNum; continue; }
        }
        genExtEmb_2V_1and2_notSameLabel_withPhase<false>(neigAddr1+indexNum1,neigAddr2+indexNum2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        if(neigV[warpIdInBlock][66]==maxRowNum){
            index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
            uint writeEle = neigAddr1+indexNum1-neighborsData;
            if(laneId==1){ writeEle = neigLen1; }
            else if(laneId==2){ writeEle = neigAddr2+indexNum2-neighborsData; }
            else if(laneId==3){ writeEle = neigLen2; }
            else if(laneId==4){ writeEle = i; }
            else if(laneId==5){ writeEle = neigV[warpIdInBlock][64]; }
            if(laneId<6){
                newEmb[(maxRowNum+index)*38+laneId] = writeEle;
            }
            newEmb[(maxRowNum+index)*38+6+laneId] = neigV[warpIdInBlock][32+laneId];
            writeEle = neigV[warpIdInBlock][65];
            if(laneId==0){
                atomicAdd(indexForIndex+256*2+66,writeEle);
            }
            __syncthreads();
            if(threadIdx.x==0){
                atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
            }
            break;
        }
        i = i + gridWarpNum;
    }
}

//this is modified
template<bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict,bool isRecord, bool useRecord>
__global__ void extEmb_2V_1src_1and2_sameLabel_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, 
    uint *edgeLabelPartition, uint *neighborsData, uint *recordPos, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, 
    uint partialRowNum, uint embLen, bool isContinue, uint maxRowNum) {

    uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM(96)
    __syncthreads();
    if(isContinue){
        uint writeEle = laneId<6?newEmb[(maxRowNum+i)*38+laneId]:0;
        neigV[warpIdInBlock][laneId+64] = writeEle; 
        neigV[warpIdInBlock][32+laneId] = newEmb[(maxRowNum+i)*38+6+laneId];
        i = neigV[warpIdInBlock][64+4];
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][64+0];
        uint neigLen1 = neighborsData[64+1];
        uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][64+2];
        uint neigLen2 = neighborsData[64+3];
        if(isExt1and2Restrict){
            genExtEmb_2V_1src_1and2_restrict_withPhase<true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        }else{
            genExtEmb_2V_1src_1and2_noRestrict_sameLabel_evalEq_withPhase<true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        }
        i = i + gridWarpNum;
    }
    while (i < partialRowNum) {
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        j = __ballot_sync(0xffffffff,j);
        if(j<0xffffffff) { i=i+gridWarpNum; continue; }

        uint svid = indexForIndex[256*2+1+1+4];
        svid = neigV[warpIdInBlock][svid];
        uint lowerLimit = 0, upperLimit, predicate = 0, index;
        uint len = (intervalNum+31)&0xffffffe0;
        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid>=lowerLimit && svid<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        uint baseNum = edgeLabelPartition[index];
        uint *neigAddr = neighborsData+baseNum;
        uint neigLen = edgeLabelPartition[index+1]-baseNum;
        uint indexNum = 0;
        if(useRecord){
            uint distance = recordPos[index*2];
            uint tmpNeigLen = recordPos[index*2+1];
            if(distance==0 && tmpNeigLen==0){
                uint evidlabel = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen;
                FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evidlabel)
                neigLen = upperLimit - lowerLimit;
                if (neigLen == 0) {
                    if (isRecord) { if (laneId == 0) { recordPos[index*2] = 1; recordPos[index*2+1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen > VBLOCKSIZE+2 && neigLen <= 32 * VBLOCKSIZE + 32) { indexNum = (neigLen+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }
                neigAddr = neigAddr + lowerLimit;
                neigLen = neigLen - indexNum;
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen; } }
            }else if(distance==1 && tmpNeigLen==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr = neigAddr+distance;
                neigLen = tmpNeigLen;
            }
        }else {
            uint evidlabel = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen;
            FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evidlabel)
            neigLen = upperLimit - lowerLimit;
            if (neigLen == 0) {
                if (isRecord) { if (laneId == 0) { recordPos[index*2] = 1; recordPos[index*2+1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen > VBLOCKSIZE+2 && neigLen <= 32 * VBLOCKSIZE + 32) { indexNum = (neigLen+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen > 32 * VBLOCKSIZE + 32) { indexNum = 32; }
            neigAddr = neigAddr + lowerLimit;
            neigLen = neigLen - indexNum;
            if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen; } }
        }
        uint neigLen1 = neigLen, neigLen2 = neigLen;
        if(isExt1Restrict){
            uint lessvid = indexForIndex[2*256+1+1+2];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum == 0) { neigLen1 = findLessVPosNoIndex(neigAddr, neigLen, lessvid, laneId); }
            else { neigLen1 = findLessVPosWithIndex(neigAddr, neigLen, lessvid, laneId, indexNum); }
            if (neigLen1 == 0) { 
                i=i+gridWarpNum;
                continue; 
            }
        }
        if(isExt2Restrict){
            uint lessvid = indexForIndex[2*256+1+1+3];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum == 0) { neigLen2 = findLessVPosNoIndex(neigAddr, neigLen, lessvid, laneId); }
            else { neigLen2 = findLessVPosWithIndex(neigAddr, neigLen, lessvid, laneId, indexNum); }
            if (neigLen2 == 0) { 
                i=i+gridWarpNum;
                continue; 
            }
        }

        if(isExt1and2Restrict){
            if(isContinue){
                genExtEmb_2V_1src_1and2_restrict_withPhase<true>(neigAddr+indexNum,neigAddr+indexNum,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
            }else{
                genExtEmb_2V_1src_1and2_restrict_withPhase<false>(neigAddr+indexNum,neigAddr+indexNum,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
            }
        }else{
            if(isContinue){
                genExtEmb_2V_1src_1and2_noRestrict_sameLabel_evalEq_withPhase<true>(neigAddr+indexNum,neigAddr+indexNum,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
            }else{
                genExtEmb_2V_1src_1and2_noRestrict_sameLabel_evalEq_withPhase<false>(neigAddr+indexNum,neigAddr+indexNum,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
            }
        }
        if(neigV[warpIdInBlock][66]==maxRowNum){
            index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
            uint writeEle = neigAddr+indexNum-neighborsData;
            if(laneId==1){ writeEle = neigLen1; }
            else if(laneId==2){ writeEle = neigAddr+indexNum-neighborsData; }
            else if(laneId==3){ writeEle = neigLen2; }
            else if(laneId==4){ writeEle = i; }
            else if(laneId==5){ writeEle = neigV[warpIdInBlock][64]; }
            if(laneId<6){
                newEmb[(maxRowNum+index)*38+laneId] = writeEle;
            }
            newEmb[(maxRowNum+index)*38+6+laneId] = neigV[warpIdInBlock][32+laneId];
            writeEle = neigV[warpIdInBlock][65];
            if(laneId==0){
                atomicAdd(indexForIndex+256*2+66,writeEle);
            }
            __syncthreads();
            if(threadIdx.x==0){
                atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
            }
            break;
        }
        i = i + gridWarpNum;
    }
}

//this is modified
template<bool isExt1Restrict,bool isExt2Restrict,bool isRecord1,bool isRecord2,bool useRecord1,bool useRecord2>
__global__ void extEmb_2V_1src_1and2_notSameLabel_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, 
    uint *neighborsData, uint *recordPos1, uint *recordPos2, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, 
    uint partialRowNum, uint embLen,bool isContinue, uint maxRowNum){

	uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM(96)
    __syncthreads();
    if(isContinue){
        uint writeEle = laneId<6?newEmb[(maxRowNum+i)*38+laneId]:0;
        neigV[warpIdInBlock][laneId+64] = writeEle; 
        neigV[warpIdInBlock][32+laneId] = newEmb[(maxRowNum+i)*38+6+laneId];
        i = neigV[warpIdInBlock][64+4];
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        uint *neigAddr1 = neighborsData+neigV[warpIdInBlock][64+0];
        uint neigLen1 = neighborsData[64+1];
        uint *neigAddr2 = neighborsData+neigV[warpIdInBlock][64+2];
        uint neigLen2 = neighborsData[64+3];
        genExtEmb_2V_1and2_notSameLabel_withPhase<true>(neigAddr1,neigAddr2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        i = i + gridWarpNum;
    }

    while (i < partialRowNum) {
        j = laneId<(embLen-2)?partialEmb[i*(embLen-2)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        j = __ballot_sync(0xffffffff,j);
        if(j<0xffffffff){ i=i+gridWarpNum; continue; }

        uint svid = indexForIndex[256*2+1+1+4];
        svid = neigV[warpIdInBlock][svid];
        uint lowerLimit = 0, upperLimit, predicate = 0, index;
        uint len = (intervalNum+31)&0xffffffe0;
        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid>=lowerLimit && svid<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        uint baseNum = edgeLabelPartition[index];
        uint *neigAddr = neighborsData+baseNum;
        uint neigLen = edgeLabelPartition[index+1]-baseNum;
        uint *neigAddr1 = neigAddr, neigLen1 = neigLen, indexNum1 = 0;
        if(useRecord1){
            uint distance = recordPos1[index*2];
            //uint *tmpNeigAddr = neigAddr + recordPos1[index*2];
            uint tmpNeigLen = recordPos1[index*2+1];
            if(distance==0 && tmpNeigLen==0){
                uint evid1label = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen;
                FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evid1label)
                neigLen1 = upperLimit - lowerLimit;
                if (neigLen1 == 0) {
                    if (isRecord1) { if (laneId == 0) { recordPos1[index*2] = 1; recordPos1[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
                neigAddr1 = neigAddr + lowerLimit;
                neigLen1 = neigLen1 - indexNum1;
                if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = lowerLimit; recordPos1[index * 2 + 1] = neigLen1; } }
            }else if(distance==1 && tmpNeigLen==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr1 = neigAddr+distance;
                neigLen1 = tmpNeigLen;
            }
        }else {
            uint evid1label = indexForIndex[2*256+1+1+0],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen;
            FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evid1label)
            neigLen1 = upperLimit - lowerLimit;
            if (neigLen1 == 0) {
                if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = 1; recordPos1[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
            neigAddr1 = neigAddr + lowerLimit;
            neigLen1 = neigLen1 - indexNum1;
            if (isRecord1) { if (laneId == 0) { recordPos1[index * 2] = lowerLimit; recordPos1[index * 2 + 1] = neigLen1; } }
        }
        if(isExt1Restrict){
            uint lessvid = indexForIndex[2*256+1+1+2];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum1 == 0) { neigLen1 = findLessVPosNoIndex(neigAddr1, neigLen1, lessvid, laneId); }
            else { neigLen1 = findLessVPosWithIndex(neigAddr1, neigLen1, lessvid, laneId, indexNum1); }
            if (neigLen1 == 0) { i=i+gridWarpNum; continue; }
        }

        uint *neigAddr2 = neigAddr, neigLen2 = neigLen, indexNum2 = 0;
        if(useRecord2){
            uint distance = recordPos2[index*2];
            //uint *tmpNeigAddr2 = neigAddr2 + recordPos2[index*2];
            uint tmpNeigLen2 = recordPos2[index*2+1];
            if(distance==0 && tmpNeigLen2==0){
                uint evid2label = indexForIndex[2*256+1+1+1],found,lessThan,greatThan;
                //evid2 is always > evid1label
                lowerLimit = neigAddr1+neigLen1-neigAddr; 
                upperLimit = neigLen2;
                FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evid2label)
                neigLen2 = upperLimit - lowerLimit;
                if (neigLen2 == 0) {
                    if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = 1; recordPos2[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
                neigAddr2 = neigAddr2 + lowerLimit;
                neigLen2 = neigLen2 - indexNum2;
                if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = lowerLimit; recordPos2[index * 2 + 1] = neigLen2; } }
            }else if(distance==1 && tmpNeigLen2==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr2 = neigAddr+distance;
                neigLen2 = tmpNeigLen2;
            }
        }else {
            uint evid2label = indexForIndex[2 * 256+1+1+1],found,lessThan,greatThan;
            lowerLimit = neigAddr1+neigLen1-neigAddr; 
            upperLimit = neigLen2;
            FIND_LABEL_LIMIT(neigAddr2, lowerLimit, upperLimit, evid2label)
            neigLen2 = upperLimit - lowerLimit;
            if (neigLen2 == 0) {
                if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = 1; recordPos2[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen2 > VBLOCKSIZE+2 && neigLen2 <= 32 * VBLOCKSIZE + 32) { indexNum2 = (neigLen2+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen2 > 32 * VBLOCKSIZE + 32) { indexNum2 = 32; }
            neigAddr2 = neigAddr2 + lowerLimit;
            neigLen2 = neigLen2 - indexNum2;
            if (isRecord2) { if (laneId == 0) { recordPos2[index * 2] = lowerLimit; recordPos2[index * 2 + 1] = neigLen2; } }
        }
        if(isExt2Restrict){
            uint lessvid = indexForIndex[2*256+1+1+3];
            lessvid = neigV[warpIdInBlock][lessvid];
            if (indexNum2 == 0) { neigLen2 = findLessVPosNoIndex(neigAddr2, neigLen2, lessvid, laneId); }
            else { neigLen2 = findLessVPosWithIndex(neigAddr2, neigLen2, lessvid, laneId, indexNum2); }
            if (neigLen2 == 0) { i = i + gridWarpNum;continue; }
        }
        genExtEmb_2V_1and2_notSameLabel_withPhase<false>(neigAddr1+indexNum1,neigAddr2+indexNum2,neigLen1,neigLen2,neigV[warpIdInBlock],indexForIndex+2*256+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        if(neigV[warpIdInBlock][66]==maxRowNum){
            index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
            uint writeEle = neigAddr1+indexNum1-neighborsData;
            if(laneId==1){ writeEle = neigLen1; }
            else if(laneId==2){ writeEle = neigAddr2+indexNum2-neighborsData; }
            else if(laneId==3){ writeEle = neigLen2; }
            else if(laneId==4){ writeEle = i; }
            else if(laneId==5){ writeEle = neigV[warpIdInBlock][64]; }
            if(laneId<6){
                newEmb[(maxRowNum+index)*38+laneId] = writeEle;
            }
            newEmb[(maxRowNum+index)*38+6+laneId] = neigV[warpIdInBlock][32+laneId];
            writeEle = neigV[warpIdInBlock][65];
            if(laneId==0){
                atomicAdd(indexForIndex+256*2+66,writeEle);
            }
            __syncthreads();
            if(threadIdx.x==0){
                atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
            }
            break;
        }
        i = i + gridWarpNum;
    }
}

//this is modified
template<bool isExtRestrict,bool isRecord,bool useRecord, bool isLastPhase>
__global__ void extEmb_1V_NoHash_kernel(uint *vLabel, uint *totWriteRowNum, uint *edgeLabelPartition, 
    uint *neighborsData, uint *recordPos, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, 
    uint embLen, bool isContinue, uint maxRowNum) {

	uint laneId, gridWarpNum, warpIdInBlock, i, j;
    INIT_VARS_SHAREDMEM(64)
	__syncthreads();

    if(isContinue){
        uint writeEle = laneId<4?newEmb[(maxRowNum+i)*4+laneId]:0;
        neigV[warpIdInBlock][laneId+32] = writeEle; 
        i = neigV[warpIdInBlock][32+2];
        j = laneId<(embLen-1)?partialEmb[i*(embLen-1)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        uint *neigAddr = neighborsData+neigV[warpIdInBlock][32];
        uint neigLen = neigV[warpIdInBlock][32+1];
        genExtEmb_1V<isLastPhase,true>(neigAddr,neigLen,neigV[warpIdInBlock],indexForIndex+256*2+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        /*if(retval>0){
            index = (blockIDx.x*blockDim.x+threadIdx.x)>>5;
            uint writeEle = neigAddr-neighborsData;
            if(laneId==1){ writeEle = neigLen; }
            else if(laneId==2){ writeEle = i; }
            else if(laneId==3){ writeEle = retval; }
            if(laneId<4){
                newEmb[(maxRowNum+index)*4+laneId] = writeEle;
            }
            break;
        }*/
        i = i + gridWarpNum;
    }
    while (i < partialRowNum) {
        j = laneId<(embLen-1)?partialEmb[i*(embLen-1)+laneId]:1;
        neigV[warpIdInBlock][laneId] = j;
        j = __ballot_sync(0xffffffff,j);
        if(j<0xffffffff){ i = i + gridWarpNum; continue; }

        uint svid = indexForIndex[256*2+1+1+4];
        svid = neigV[warpIdInBlock][svid];

        uint lowerLimit = 0, upperLimit, predicate = 0, index;
        uint len = (intervalNum+31)&0xffffffe0;
        for(j=laneId;j<len;j=j+32){
            lowerLimit = indexForIndex[j];
            upperLimit = indexForIndex[j+intervalNum+1];
            upperLimit = lowerLimit+upperLimit-indexForIndex[j+intervalNum];
            predicate = j<intervalNum?(svid>=lowerLimit && svid<upperLimit):0;
            predicate = __ballot_sync(0xffffffff,predicate);
            if(predicate>0){
                index = svid-lowerLimit+indexForIndex[j+intervalNum];
                uint tmpIndex = __ffs(predicate)-1;
                index = __shfl_sync(0xffffffff,index,tmpIndex);
                break;
            }
        }
        if(predicate==0){ 
            i=i+gridWarpNum;
            continue; 
        }
        uint baseNum = edgeLabelPartition[index];
        uint *neigAddr = neighborsData+baseNum;
        uint neigLen = edgeLabelPartition[index+1]-baseNum;
        uint *neigAddr1 = neigAddr, neigLen1 = neigLen, indexNum1 = 0;
        if(useRecord){
            uint distance = recordPos[index*2];
            //uint *tmpNeigAddr = neigAddr + recordPos[index*2];
            uint tmpNeigLen = recordPos[index*2+1];
            if(distance==0 && tmpNeigLen==0){
                uint evidlabel = indexForIndex[2 * 256+1+1+0],found,lessThan,greatThan;
                lowerLimit = 0; upperLimit = neigLen;
                FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evidlabel)
                neigLen1 = upperLimit - lowerLimit;
                if (neigLen1 == 0) {
                    if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                    i = i + gridWarpNum;
                    continue;
                }
                if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
                else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
                neigAddr1 = neigAddr + lowerLimit;
                neigLen1 = neigLen1 - indexNum1;
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen1; } }
            }else if(distance==1 && tmpNeigLen==0){
                i = i+gridWarpNum;
                continue;
            }else{
                neigAddr1 = neigAddr+distance;
                neigLen1 = tmpNeigLen;
            }
        }else {
            uint evidlabel = indexForIndex[2 * 256+1+1+0],found,lessThan,greatThan;
            lowerLimit = 0; upperLimit = neigLen;
			FIND_LABEL_LIMIT(neigAddr, lowerLimit, upperLimit, evidlabel)
            neigLen1 = upperLimit- lowerLimit;
            if (neigLen1 == 0) {
                if (isRecord) { if (laneId == 0) { recordPos[index * 2] = 1; recordPos[index * 2 + 1] = 0; } }
                i = i + gridWarpNum;
                continue;
            }
            if (neigLen1 > VBLOCKSIZE+2 && neigLen1 <= 32 * VBLOCKSIZE + 32) { indexNum1 = (neigLen1+VBLOCKSIZE) / (VBLOCKSIZE + 1); }
            else if (neigLen1 > 32 * VBLOCKSIZE + 32) { indexNum1 = 32; }
            neigAddr1 = neigAddr + lowerLimit;
            neigLen1 = neigLen1 - indexNum1;
            if (isRecord) { if (laneId == 0) { recordPos[index * 2] = lowerLimit; recordPos[index * 2 + 1] = neigLen1; } }
        }
        if(isExtRestrict){
            uint lesspos = indexForIndex[256*2+1+1+2];
            uint lessvid = neigV[warpIdInBlock][lesspos];
            if (indexNum1 == 0) { neigLen1 = findLessVPosNoIndex(neigAddr1, neigLen1, lessvid, laneId); }
            else { neigLen1 = findLessVPosWithIndex(neigAddr1, neigLen1, lessvid, laneId, indexNum1); }
            if (neigLen1 == 0) { i = i + gridWarpNum; continue; }
        }
        genExtEmb_1V<isLastPhase,false>(neigAddr1+indexNum1,neigLen1,neigV[warpIdInBlock],indexForIndex+256*2+1+1+6,totWriteRowNum,newEmb,laneId,embLen,maxRowNum);
        if(neigV[warpIdInBlock][34]==maxRowNum){
            index = (blockIdx.x*blockDim.x+threadIdx.x)>>5;
            uint writeEle = neigAddr1+indexNum1-neighborsData;
            if(laneId==1){ writeEle = neigLen1; }
            else if(laneId==2){ writeEle = i; }
            else if(laneId==3){ writeEle = neigV[warpIdInBlock][32]; }
            if(laneId<4){
                newEmb[(maxRowNum+index)*4+laneId] = writeEle;
            }
            writeEle = neigV[warpIdInBlock][33];
            if(laneId==0){
                atomicAdd(indexForIndex+256*2+66,writeEle);
            }
            __syncthreads();
            if(threadIdx.x==0){
                atomicAdd(totWriteRowNum+1,indexForIndex[256*2+66]);
            }
            break;
        }
        i = i + gridWarpNum;
    }
}

extern int GPU_SM_NUM;

#define INITEMB_2V(isExtRestrict,isRecord) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,initEmb_2V_NoHash_kernel<isExtRestrict,isRecord>,WARPPERBLOCK * 32, 0);\
    initEmb_2V_NoHash_kernel<isExtRestrict,isRecord><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos,newEmb,intervalNum,totSrcNum,svidlabel,evidlabel,isContinue,maxRowNum);


#define INITEMB_3V(isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isExt1and2SameLabel,isRecord1,isRecord2) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,initEmb_3V_1and2_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isExt1and2SameLabel,isRecord1,isRecord2>,WARPPERBLOCK * 32, 0);\
    initEmb_3V_1and2_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isExt1and2SameLabel,isRecord1,isRecord2><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos1,recordPos2,newEmb,intervalNum,totSrcNum,svidlabel,evid1label,evid2label,isContinue,maxRowNum);

#define EXTEMB_1V(isExtRestrict,isRecord,useRecord,isLastPhase) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,extEmb_1V_NoHash_kernel<isExtRestrict,isRecord,useRecord,isLastPhase>,WARPPERBLOCK*32,0);\
    extEmb_1V_NoHash_kernel<isExtRestrict,isRecord,useRecord,isLastPhase><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos,auxArray,newEmb,partialEmb,intervalNum,partialRowNum,embLen,isContinue,maxRowNum);\

#define EXTEMB_2V_2S_1L(isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,extEmb_2V_2src_1and2_sameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord>,WARPPERBLOCK*32,0);\
    extEmb_2V_2src_1and2_sameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos,auxArray,newEmb,partialEmb,intervalNum,partialRowNum,embLen,isContinue,maxRowNum);

#define EXTEMB_2V_1S_2L(isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,extEmb_2V_1src_1and2_notSameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2>,WARPPERBLOCK*32,0);\
    extEmb_2V_1src_1and2_notSameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos1,recordPos2,auxArray,newEmb,partialEmb,intervalNum,partialRowNum,embLen,isContinue,maxRowNum);

#define EXTEMB_2V_1S_1L(isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,extEmb_2V_1src_1and2_sameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord>,WARPPERBLOCK*32,0);\
    extEmb_2V_1src_1and2_sameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isExt1and2Restrict,isRecord,useRecord><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos,auxArray,newEmb,partialEmb,intervalNum,partialRowNum,embLen,isContinue,maxRowNum);

#define EXTEMB_2V_2S_2L(isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2) \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,extEmb_2V_2src_1and2_notSameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2>,WARPPERBLOCK*32,0);\
    extEmb_2V_2src_1and2_notSameLabel_NoHash_kernel<isExt1Restrict,isExt2Restrict,isRecord1,isRecord2,useRecord1,useRecord2><<<numBlocks*GPU_SM_NUM,WARPPERBLOCK*32>>>(vLabel,writeRowNum,edgeLabelPartition,neighborsData,recordPos1,recordPos2,auxArray,newEmb,partialEmb,intervalNum,partialRowNum,embLen,isContinue,maxRowNum);


void initEmb_3V_1and2_NoHash(uint *vLabel,  uint *writeRowNum, uint *edgeLabelPartition, uint *neighborsData, uint *recordPos1,
    uint *recordPos2, uint *newEmb, uint intervalNum, uint totSrcNum, uint svidlabel,uint evid1label,uint evid2label,
    bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict, bool isExt1and2SameLabel, bool isRecord1, bool isRecord2,
    bool isContinue, uint maxRowNum){
    int numBlocks;
    if(isExt1and2SameLabel && isRecord1){
        if(isExt1Restrict && isExt1and2Restrict){   INITEMB_3V(true,false,true,true,true,false)    }
        else if(isExt1Restrict){                    INITEMB_3V(true,false,false,true,true,false)   }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,true,true,false)   }
        else if(isExt1and2Restrict){                INITEMB_3V(false,false,true,true,true,false)   }
        else{                                       INITEMB_3V(false,false,false,true,true,false)  }
    }else if(isExt1and2SameLabel && !isRecord1){
        if(isExt1Restrict && isExt1and2Restrict){   INITEMB_3V(true,false,true,true,false,false)   }
        else if(isExt1Restrict){                    INITEMB_3V(true,false,false,true,false,false)  }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,true,false,false)  }
        else if(isExt1and2Restrict){                INITEMB_3V(false,false,true,true,false,false)  }
        else{                                       INITEMB_3V(false,false,false,true,false,false) }
    }else if(!isExt1and2SameLabel && isRecord1 && isRecord2){
        if(isExt1Restrict){                         INITEMB_3V(true,false,false,false,true,true)   }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,false,true,true)   }
        else{                                       INITEMB_3V(false,false,false,false,true,true)  }
    }else if(!isExt1and2SameLabel && isRecord1){
        if(isExt1Restrict){                         INITEMB_3V(true,false,false,false,true,false)  }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,false,true,false)  }
        else{                                       INITEMB_3V(false,false,false,false,true,false) }
    }else if(!isExt1and2SameLabel && isRecord2){
        if(isExt1Restrict){                         INITEMB_3V(true,false,false,false,false,true)  }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,false,false,true)  }
        else{                                       INITEMB_3V(false,false,false,false,false,true) }
    }else if(!isExt1and2SameLabel){
        if(isExt1Restrict){                         INITEMB_3V(true,false,false,false,false,false) }
        else if(isExt2Restrict){                    INITEMB_3V(false,true,false,false,false,false) }
        else{                                       INITEMB_3V(false,false,false,false,false,false)}
    }
}


void initEmb_2V_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint *neighborsData, uint *recordPos,
    uint *newEmb, uint intervalNum, uint totSrcNum, uint svidlabel, uint evidlabel, uint isExtRestrict, uint isRecord,
    bool isContinue, uint maxRowNum){
    
    int numBlocks;
    if(isRecord && isExtRestrict) {       INITEMB_2V(true,true)  }
    else if(isRecord && !isExtRestrict){  INITEMB_2V(false,true) }
    else if(!isRecord && isExtRestrict){  INITEMB_2V(true,false) }
    else{                                 INITEMB_2V(false,false)}                             
}

void extEmb_2V_2src_1and2_sameLabel_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint *neighborsData,
    uint *recordPos, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, uint embLen,
    bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict,bool isRecord, bool useRecord, bool isContinue, uint maxRowNum){

    int numBlocks;
    if (isExt1Restrict) {
        if (isExt1and2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(true, false, true, true,true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(true, false, true, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(true, false, true, false, true) }
            else {                              EXTEMB_2V_2S_1L(true, false, true, false, false) }
        } else if (isExt2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(true, true, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(true, true, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(true, true, false, false, true) }
            else {                              EXTEMB_2V_2S_1L(true, true, false, false, false) }
        } else {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(true, false, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(true, false, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(true, false, false, false, true) }
            else {                              EXTEMB_2V_2S_1L(true, false, false, false, false) }
        }
    } else {
        if (isExt1and2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(false, false, true, true,true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(false, false, true, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(false, false, true, false, true) }
            else {                              EXTEMB_2V_2S_1L(false, false, true, false, false) }
        } else if (isExt2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(false, true, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(false, true, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(false, true, false, false, true) }
            else {                              EXTEMB_2V_2S_1L(false, true, false, false, false) }
        } else {
            if(isRecord && useRecord){        EXTEMB_2V_2S_1L(false, false, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_2S_1L(false, false, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_2S_1L(false, false, false, false, true) }
            else {                              EXTEMB_2V_2S_1L(false, false, false, false, false) }
        }
    }
}

void extEmb_2V_2src_1and2_notSameLabel_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint*neighborsData, 
    uint *recordPos1, uint *recordPos2, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, 
    uint embLen, bool isExt1Restrict, bool isExt2Restrict,bool isRecord1,bool isRecord2,bool useRecord1,bool useRecord2,
    bool isContinue, uint maxRowNum){

    int numBlocks;
    if(isExt1Restrict && isExt2Restrict){
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,true,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,true,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,true,true,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,true,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,true,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,true,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,true,true,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,true,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,true,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,true,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,true,false,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,true,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,true,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,true,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,true,false,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,true,false,false,false,false) }
        }
    } else if(isExt1Restrict && !isExt2Restrict){
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,false,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,false,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,false,true,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,false,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,false,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,false,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,false,true,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,false,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,false,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,false,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,false,false,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,false,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(true,false,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(true,false,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(true,false,false,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(true,false,false,false,false,false) }
        }
    }else if(!isExt1Restrict && isExt2Restrict){
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,true,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,true,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,true,true,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,true,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,true,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,true,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,true,true,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,true,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,true,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,true,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,true,false,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,true,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,true,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,true,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,true,false,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,true,false,false,false,false) }
        }
    }else{
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,false,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,false,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,false,true,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,false,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,false,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,false,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,false,true,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,false,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,false,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,false,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,false,false,true,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,false,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_2S_2L(false,false,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_2S_2L(false,false,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_2S_2L(false,false,false,false,false,true) }
            else {                               EXTEMB_2V_2S_2L(false,false,false,false,false,false) }
        }
    }
}


void extEmb_2V_1src_1and2_sameLabel_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint*neighborsData, 
    uint *recordPos, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, uint embLen, 
    bool isExt1Restrict, bool isExt2Restrict, bool isExt1and2Restrict,bool isRecord, bool useRecord, bool isContinue, uint maxRowNum){

    int numBlocks;
    if (isExt1Restrict) {
        if (isExt1and2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(true, false, true, true,true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(true, false, true, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(true, false, true, false, true) }
            else {                            EXTEMB_2V_1S_1L(true, false, true, false, false) }
        } else if (isExt2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(true, true, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(true, true, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(true, true, false, false, true) }
            else {                            EXTEMB_2V_1S_1L(true, true, false, false, false) }
        } else {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(true, false, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(true, false, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(true, false, false, false, true) }
            else {                            EXTEMB_2V_1S_1L(true, false, false, false, false) }
        }
    } else {
        if (isExt1and2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(false, false, true, true,true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(false, false, true, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(false, false, true, false, true) }
            else {                            EXTEMB_2V_1S_1L(false, false, true, false, false) }
        } else if (isExt2Restrict) {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(false, true, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(false, true, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(false, true, false, false, true) }
            else {                            EXTEMB_2V_1S_1L(false, true, false, false, false) }
        } else {
            if(isRecord && useRecord){        EXTEMB_2V_1S_1L(false, false, false, true, true) }
            else if(isRecord && !useRecord) { EXTEMB_2V_1S_1L(false, false, false, true, false) }
            else if(!isRecord && useRecord) { EXTEMB_2V_1S_1L(false, false, false, false, true) }
            else {                            EXTEMB_2V_1S_1L(false, false, false, false, false) }
        }
    }
}


void extEmb_2V_1src_1and2_notSameLabel_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint *neighborsData, 
    uint *recordPos1, uint *recordPos2, uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, 
    uint embLen, bool isExt1Restrict, bool isExt2Restrict, bool isRecord1, bool isRecord2, bool useRecord1, bool useRecord2,
    bool isContinue, uint maxRowNum){

    int numBlocks;
    if (isExt1Restrict && isExt2Restrict) {
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,true,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,true,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,true,true,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,true,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,true,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,true,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,true,true,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,true,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,true,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,true,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,true,false,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,true,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,true,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,true,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,true,false,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,true,false,false,false,false) }
        }
    } else if(isExt1Restrict && !isExt2Restrict){
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,false,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,false,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,false,true,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,false,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,false,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,false,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,false,true,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,false,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,false,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,false,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,false,false,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,false,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(true,false,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(true,false,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(true,false,false,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(true,false,false,false,false,false) }
        }
    }else if(!isExt1Restrict && isExt2Restrict){
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,true,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,true,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,true,true,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,true,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,true,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,true,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,true,true,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,true,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,true,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,true,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,true,false,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,true,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,true,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,true,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,true,false,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,true,false,false,false,false) }
        }
    }else{
        if(isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,false,true,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,false,true,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,false,true,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,false,true,true,false,false) }
        } else if(isRecord1 && !isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,false,true,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,false,true,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,false,true,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,false,true,false,false,false) }
        }else if(!isRecord1 && isRecord2){
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,false,false,true,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,false,false,true,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,false,false,true,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,false,false,true,false,false) }
        }else{
            if (useRecord1 && useRecord2) {      EXTEMB_2V_1S_2L(false,false,false,false,true,true) }
            else if(useRecord1 && !useRecord2) { EXTEMB_2V_1S_2L(false,false,false,false,true,false) }
            else if(!useRecord1 && useRecord2) { EXTEMB_2V_1S_2L(false,false,false,false,false,true) }
            else {                               EXTEMB_2V_1S_2L(false,false,false,false,false,false) }
        }
    }
}


void extEmb_1V_NoHash(uint *vLabel, uint *writeRowNum, uint *edgeLabelPartition, uint *neighborsData, uint *recordPos, 
    uint *auxArray, uint *newEmb, uint *partialEmb, uint intervalNum, uint partialRowNum, uint embLen, bool isExtRestrict, 
    bool isRecord, bool useRecord, bool isLastPhase, bool isContinue, uint maxRowNum){

    int numBlocks;
    if(isLastPhase){
        if(isExtRestrict){
            if(isRecord){
                if(useRecord){ EXTEMB_1V(true,true,true,true) }
                else{          EXTEMB_1V(true,true,false,true)}
            }else{
                if(useRecord){ EXTEMB_1V(true,false,true,true) }
                else{          EXTEMB_1V(true,false,false,true)}
            }
        }else{
            if(isRecord){
                if(useRecord){ EXTEMB_1V(false,true,true,true) }
                else{          EXTEMB_1V(false,true,false,true)}
            }else{
                if(useRecord){ EXTEMB_1V(false,false,true,true)}
                else{          EXTEMB_1V(false,false,false,true)}
            }
        }
    }else{
        if(isExtRestrict){
            if(isRecord){
                if(useRecord){ EXTEMB_1V(true,true,true,false) }
                else{          EXTEMB_1V(true,true,false,false)}
            }else{
                if(useRecord){ EXTEMB_1V(true,false,true,false) }
                else{          EXTEMB_1V(true,false,false,false)}
            }
        }else{
            if(isRecord){
                if(useRecord){ EXTEMB_1V(false,true,true,false) }
                else{          EXTEMB_1V(false,true,false,false)}
            }else{
                if(useRecord){ EXTEMB_1V(false,false,true,false)}
                else{          EXTEMB_1V(false,false,false,false)}
            }
        }
    }

}
